#include "hip/hip_runtime.h"
#include "NN.cuh"

#define THREADS_PER_BLOCK 32

// Device functions:

__device__ float d_relu(float val)
{
    return val > 0.0f ? val : 0.0f;
}

__device__ float d_derive_relu(float val)
{
    return val > 0.0f ? 1.0f : 0.0f;
}

__device__ float d_sigmoid(float val)
{
    return (1.0 / (1.0 + exp(-val)));
}

__device__ float d_derive_sigmoid(float val)
{
    return (val) * (1.0 - val);
}

__device__ float d_tanh(float val)
{
    return ((exp(val) - exp(-val)) / (exp(val) + exp(-val)));
}

__device__ float d_derive_tanh(float val)
{
    return (1 - pow(val, 2.0));
}

__device__ float d_mse_cost(float n_val, float y_val)
{
    return pow((n_val - y_val), 2.0);
}

__device__ float d_derive_mse_cost(float n_val, float y_val)
{
    return 2.0f * (n_val - y_val);
}

__device__ float d_cross_entropy_cost(float n_val, float y_val)
{
    return (float)((y_val * log(n_val)) + ((1.0 - y_val) * log(1.0 - n_val)));
}

__device__ float d_derive_cross_entropy_cost(float n_val, float y_val)
{
    return (n_val - y_val);
}

// Kernel functions:

__global__ void k_set_arr(float *arr, int cnt, float val)
{
    int tid = blockIdx.x * blockDim.x + threadIdx.x;

    if (tid < cnt)
    {
        arr[tid] = val;
    }
}

__global__ void k_dot(float *n_arr, float *w_arr, float *nxt_n_arr, int n_cnt, int nxt_n_cnt)
{
    __shared__ float temp[THREADS_PER_BLOCK];
    memset(temp, 0, THREADS_PER_BLOCK * sizeof(float));

    int tid = blockIdx.x * blockDim.x + threadIdx.x;

    int w_cnt = n_cnt * nxt_n_cnt;

    int n_idx = tid % n_cnt;
    int nxt_n_idx = tid / n_cnt;
    int w_idx = tid;

    if (tid < w_cnt)
    {
        temp[threadIdx.x] = n_arr[n_idx] * w_arr[w_idx];
    }

    __syncthreads();

    if (threadIdx.x == 0) // threadIdx MUST be 0 for below logic to work!
    {
        /*
        The goal here is to try to minimize atomic adds. If the neuron count is
        greater than or equal to the threads per block, a maximum of 2 atomic adds
        is necessary for this block. However, most of the time we can get away with just 1.

        If the threads per block is greater than the neuron count, we just play it safe
        and incur an atomic add for each thread in the block.
        */

        int lower_idx = tid / n_cnt;
        int upper_idx = (tid + THREADS_PER_BLOCK) / n_cnt;

        if (n_cnt >= THREADS_PER_BLOCK)
        {
            if (lower_idx == upper_idx)
            {
                float sum = 0.0f;

                for (int i = 0; i < THREADS_PER_BLOCK; i++)
                {
                    sum += temp[i];
                }

                atomicAdd(&nxt_n_arr[lower_idx], sum);
            }
            else
            {
                float sums[2] = {0.0f, 0.0f};

                for (int i = 0; i < THREADS_PER_BLOCK; i++)
                {
                    if ((tid + i) / n_cnt == lower_idx)
                    {
                        sums[0] += temp[i];
                    }
                    else
                    {
                        sums[1] += temp[i];
                    }
                }

                atomicAdd(&nxt_n_arr[lower_idx], sums[0]);
                if (upper_idx < n_cnt)
                {
                    atomicAdd(&nxt_n_arr[upper_idx], sums[1]);
                }
            }
        }
        else
        {
            for (int i = 0; i < THREADS_PER_BLOCK; i++)
            {
                atomicAdd(&nxt_n_arr[(tid + i) / n_cnt], temp[i]);
            }
        }
    }
}

__global__ void k_add_bias(float *b_arr, float *nxt_n_arr, int nxt_n_cnt)
{
    int tid = blockIdx.x * blockDim.x + threadIdx.x;

    if (tid < nxt_n_cnt)
    {
        nxt_n_arr[tid] += b_arr[tid];
    }
}

__global__ void k_activate(float *n_arr, int n_cnt, ActivationFunctionId activation_func_id)
{
    int tid = blockIdx.x * blockDim.x + threadIdx.x;

    if (tid < n_cnt)
    {
        switch (activation_func_id)
        {
        case ReLU:
            n_arr[tid] = d_relu(n_arr[tid]);
            break;
        case Sigmoid:
            n_arr[tid] = d_sigmoid(n_arr[tid]);
            break;
        case Tanh:
            n_arr[tid] = d_tanh(n_arr[tid]);
            break;
        default:
            // None
            break;
        }
    }
}

__global__ void k_cost(float *n_arr, float *y_arr, float *cost, int n_cnt, CostFunctionId cost_func_id)
{
    __shared__ float temp[THREADS_PER_BLOCK];
    memset(temp, 0, THREADS_PER_BLOCK * sizeof(float));

    int tid = blockIdx.x * blockDim.x + threadIdx.x;

    if (tid < n_cnt)
    {
        switch (cost_func_id)
        {
        case MSE:
            temp[threadIdx.x] = d_mse_cost(n_arr[tid], y_arr[tid]);
            break;
        case CrossEntropy:
            temp[threadIdx.x] = d_cross_entropy_cost(n_arr[tid], y_arr[tid]);
            break;
        default:
            break;
        }
    }

    __syncthreads();

    if (threadIdx.x == 0)
    {
        float sum = 0.0f;

        for (int i = 0; i < THREADS_PER_BLOCK; i++)
        {
            sum += temp[i];
        }

        atomicAdd(cost, sum);
    }
}

__global__ void k_derive_cost(float *n_arr, float *y_arr, float *agg_arr, int n_cnt, CostFunctionId cost_func_id)
{
    int tid = blockIdx.x * blockDim.x + threadIdx.x;

    if (tid < n_cnt)
    {
        switch (cost_func_id)
        {
        case MSE:
            agg_arr[tid] *= d_derive_mse_cost(n_arr[tid], y_arr[tid]);
            break;
        case CrossEntropy:
            agg_arr[tid] *= d_derive_cross_entropy_cost(n_arr[tid], y_arr[tid]);
            break;
        default:
            break;
        }
    }
}

__global__ void k_derive_activation(float *n_arr, float *agg_arr, int n_cnt, ActivationFunctionId activation_func_id)
{
    int tid = blockIdx.x * blockDim.x + threadIdx.x;

    if (tid < n_cnt)
    {
        switch (activation_func_id)
        {
        case ReLU:
            agg_arr[tid] *= d_derive_relu(n_arr[tid]);
            break;
        case Sigmoid:
            agg_arr[tid] *= d_derive_sigmoid(n_arr[tid]);
            break;
        case Tanh:
            agg_arr[tid] *= d_derive_tanh(n_arr[tid]);
            break;
        default:
            // None
            break;
        }
    }
}

__global__ void k_derive_z_and_increment_weight_derivative(float *agg_arr, float *n_arr, float *dw_arr, int n_cnt, int prv_n_cnt)
{
    int tid = blockIdx.x * blockDim.x + threadIdx.x;

    int w_cnt = n_cnt * prv_n_cnt;

    int prv_n_idx = tid % prv_n_cnt;
    int n_idx = tid / prv_n_cnt;
    int w_idx = n_idx * prv_n_cnt + prv_n_idx;

    if (w_idx < w_cnt)
    {
        dw_arr[w_idx] += (agg_arr[n_idx] * n_arr[prv_n_idx]);
    }
}

__global__ void k_derive_z_and_increment_bias_derivative(float *agg_arr, float *db_arr, int n_cnt)
{
    int tid = blockIdx.x * blockDim.x + threadIdx.x;

    if (tid < n_cnt)
    {
        db_arr[tid] += (agg_arr[tid]);
    }
}

__global__ void k_derive_z_and_aggregate_derivatives(float *w_arr, float *agg_arr, float *temp_agg_arr, int prv_n_cnt, int n_cnt)
{
    __shared__ float temp[THREADS_PER_BLOCK];
    memset(temp, 0, THREADS_PER_BLOCK * sizeof(float));

    int tid = blockIdx.x * blockDim.x + threadIdx.x;

    int w_cnt = prv_n_cnt * n_cnt;

    // Transpose the weights "matrix".
    int n_idx = tid % n_cnt;
    int prv_n_idx = tid / n_cnt;
    int w_idx = n_idx * prv_n_cnt + prv_n_idx;

    if (w_idx < w_cnt)
    {
        temp[threadIdx.x] = (agg_arr[n_idx] * w_arr[w_idx]);
    }

    __syncthreads();

    if (threadIdx.x == 0) // threadIdx MUST be 0 for below logic to work!
    {
        /*
        The goal here is to try to minimize atomic adds. If the neuron count is
        greater than or equal to the threads per block, a maximum of 2 atomic adds
        is necessary for this block. However, most of the time we can get away with just 1.

        If the threads per block is greater than the neuron count, we just play it safe
        and incur an atomic add for each thread in the block.
        */

        int lower_idx = tid / n_cnt;
        int upper_idx = (tid + THREADS_PER_BLOCK) / n_cnt;

        if (n_cnt >= THREADS_PER_BLOCK)
        {
            if (lower_idx == upper_idx)
            {
                float sum = 0.0f;

                for (int i = 0; i < THREADS_PER_BLOCK; i++)
                {
                    sum += temp[i];
                }
                atomicAdd(&temp_agg_arr[lower_idx], sum);
            }
            else
            {
                float sums[2] = {0.0f, 0.0f};

                for (int i = 0; i < THREADS_PER_BLOCK; i++)
                {
                    if ((tid + i) / n_cnt == lower_idx)
                    {
                        sums[0] += temp[i];
                    }
                    else
                    {
                        sums[1] += temp[i];
                    }
                }

                atomicAdd(&temp_agg_arr[lower_idx], sums[0]);
                if (upper_idx < n_cnt)
                {
                    atomicAdd(&temp_agg_arr[upper_idx], sums[1]);
                }
            }
        }
        else
        {
            for (int i = 0; i < THREADS_PER_BLOCK; i++)
            {
                atomicAdd(&temp_agg_arr[(tid + i) / n_cnt], temp[i]);
            }
        }
    }
}

__global__ void k_adjust_weight(float *w_arr, float *dw_arr, int batch_size, float learning_rate, int cnt)
{
    int tid = blockIdx.x * blockDim.x + threadIdx.x;

    if (tid < cnt)
    {
        w_arr[tid] -= ((dw_arr[tid] * learning_rate) / (float)batch_size);
        dw_arr[tid] = 0.0f;
    }
}

__global__ void k_adjust_bias(float *b_arr, float *db_arr, int batch_size, float learning_rate, int cnt)
{
    int tid = blockIdx.x * blockDim.x + threadIdx.x;

    if (tid < cnt)
    {
        b_arr[tid] -= ((db_arr[tid] * learning_rate) / (float)batch_size);
        db_arr[tid] = 0.0f;
    }
}

// Static functions:

void NN::write_csv_header(FILE *csv_file_ptr)
{
    fprintf(csv_file_ptr, "epoch,cost,accuracy,correct_cnt,total_cnt\n");
}

void NN::write_to_csv(FILE *csv_file_ptr, int epoch, ProgressReport rpt)
{
    fprintf(csv_file_ptr, "%d,%f,%f,%d,%d\n", epoch, rpt.cost, (rpt.crct_cnt / rpt.tot_cnt) * 100.0f, rpt.crct_cnt, rpt.tot_cnt);
}

// Member functions:

NN::NN(std::vector<int> layer_config, ActivationFunctionId hidden_layer_activation_func_id,
       ActivationFunctionId output_layer_activation_func_id, CostFunctionId cost_func_id, float learning_rate)
{

    int lyr_cnt = layer_config.size();

    // Leave input neurons NULL for now!
    this->neurons.push_back(nullptr);

    for (int lyr_idx = 0; lyr_idx < lyr_cnt - 1; lyr_idx++)
    {
        int n_cnt = layer_config[lyr_idx];
        int nxt_n_cnt = layer_config[lyr_idx + 1];

        Tensor *n = new Tensor(1, nxt_n_cnt, Gpu);
        n->set_all(0.0f);
        this->neurons.push_back(n);

        Tensor *w = new Tensor(nxt_n_cnt, n_cnt, Gpu);
        w->set_all_rand(1.0f / sqrt(n_cnt)); // Xavier initialization!
        this->weights.push_back(w);

        Tensor *b = new Tensor(nxt_n_cnt, 1, Gpu);
        b->set_all(0.0f);
        this->biases.push_back(b);

        Tensor *dw = new Tensor(nxt_n_cnt, n_cnt, Gpu);
        dw->set_all(0.0f);
        this->weight_derivatives.push_back(dw);

        Tensor *db = new Tensor(nxt_n_cnt, 1, Gpu);
        db->set_all(0.0f);
        this->bias_derivatives.push_back(db);
    }

    this->hidden_layer_activation_func_id = hidden_layer_activation_func_id;
    this->output_layer_activation_func_id = output_layer_activation_func_id;

    this->cost_func_id = cost_func_id;

    this->learning_rate = learning_rate;
}

NN::~NN()
{
    int lyr_cnt = this->neurons.size();

    // Do not free input neurons since we do not own the Tensor!
    this->neurons[0] = nullptr;

    for (int lyr_idx = 0; lyr_idx < lyr_cnt - 1; lyr_idx++)
    {
        delete this->neurons[lyr_idx + 1];
        delete this->weights[lyr_idx];
        delete this->biases[lyr_idx];
        delete this->weight_derivatives[lyr_idx];
        delete this->bias_derivatives[lyr_idx];
    }
}

void NN::feed_forward(Tensor *x)
{
    x->translate(Gpu);
    this->neurons[0] = x;

    int lyr_cnt = this->neurons.size();
    int lst_lyr_idx = lyr_cnt - 1;

    for (int lyr_idx = 0; lyr_idx < lst_lyr_idx; lyr_idx++)
    {
        Tensor *n = this->neurons[lyr_idx];
        Tensor *w = this->weights[lyr_idx];
        Tensor *b = this->biases[lyr_idx];
        Tensor *nxt_n = this->neurons[lyr_idx + 1];

        int n_cnt = w->get_col_cnt();
        int nxt_n_cnt = w->get_row_cnt();

        ActivationFunctionId activation_func_id = (lyr_idx == lst_lyr_idx - 1) ? this->output_layer_activation_func_id : this->hidden_layer_activation_func_id;

        // Need to reset neurons before we do anything:
        {
            int threads_per_block(THREADS_PER_BLOCK);
            int num_blocks((nxt_n_cnt / threads_per_block) + 1);
            k_set_arr<<<num_blocks, threads_per_block>>>(nxt_n->get_arr(Gpu), nxt_n_cnt, 0.0f);
        }

        // Dot product:
        {
            int threads_per_block(THREADS_PER_BLOCK);
            int num_blocks(((n_cnt * nxt_n_cnt) / threads_per_block) + 1);
            k_dot<<<num_blocks, threads_per_block>>>(n->get_arr(Gpu), w->get_arr(Gpu),
                                                     nxt_n->get_arr(Gpu), n_cnt, nxt_n_cnt);
        }

        // Add biases:
        {
            int threads_per_block(THREADS_PER_BLOCK);
            int num_blocks((nxt_n_cnt / threads_per_block) + 1);
            k_add_bias<<<num_blocks, threads_per_block>>>(b->get_arr(Gpu), nxt_n->get_arr(Gpu),
                                                          nxt_n_cnt);
        }

        // Activate:
        {
            int threads_per_block(THREADS_PER_BLOCK);
            int num_blocks((nxt_n_cnt / threads_per_block) + 1);
            k_activate<<<num_blocks, threads_per_block>>>(nxt_n->get_arr(Gpu), nxt_n_cnt, activation_func_id);
        }
    }
}

float NN::get_cost(Tensor *y)
{
    y->translate(Gpu);

    float h_cost = 0.0f;
    float *d_cost;
    hipMalloc(&d_cost, sizeof(float));
    hipMemset(d_cost, 0, sizeof(float));

    int lyr_cnt = this->neurons.size();
    int lst_lyr_idx = lyr_cnt - 1;

    Tensor *lst_lyr_n = this->neurons[lst_lyr_idx];

    int lst_lyr_n_cnt = lst_lyr_n->get_col_cnt();

    {
        int threads_per_block(THREADS_PER_BLOCK);
        int num_blocks((lst_lyr_n_cnt / threads_per_block) + 1);

        k_cost<<<num_blocks, threads_per_block>>>(lst_lyr_n->get_arr(Gpu), y->get_arr(Gpu),
                                                  d_cost, lst_lyr_n_cnt, this->cost_func_id);
    }

    hipMemcpy(&h_cost, d_cost, sizeof(float), hipMemcpyDeviceToHost);
    hipFree(d_cost);

    return h_cost;
}

void NN::back_propagate(Tensor *y)
{
    y->translate(Gpu);

    int lyr_cnt = this->neurons.size();
    int lst_lyr_idx = lyr_cnt - 1;
    int lst_lyr_n_cnt = this->neurons[lst_lyr_idx]->get_col_cnt();

    Tensor *agg = new Tensor(1, lst_lyr_n_cnt, Gpu);
    agg->set_all(1.0f);

    // Derive cost (activation):
    {
        int threads_per_block(THREADS_PER_BLOCK);
        int num_blocks((lst_lyr_n_cnt / threads_per_block) + 1);
        k_derive_cost<<<num_blocks, threads_per_block>>>(this->neurons[lst_lyr_idx]->get_arr(Gpu),
                                                         y->get_arr(Gpu), agg->get_arr(Gpu), lst_lyr_n_cnt, this->cost_func_id);
    }

    for (int lyr_idx = lst_lyr_idx; lyr_idx > 0; lyr_idx--)
    {
        Tensor *n = this->neurons[lyr_idx];
        Tensor *prv_n = this->neurons[lyr_idx - 1];
        Tensor *prv_w = this->weights[lyr_idx - 1];
        Tensor *prv_b = this->biases[lyr_idx - 1];
        Tensor *prv_dw = this->weight_derivatives[lyr_idx - 1];
        Tensor *prv_db = this->bias_derivatives[lyr_idx - 1];

        int n_cnt = prv_w->get_row_cnt();
        int prv_n_cnt = prv_w->get_col_cnt();

        ActivationFunctionId activation_func_id = (lyr_idx == lst_lyr_idx) ? this->output_layer_activation_func_id : this->hidden_layer_activation_func_id;

        // Derive activation (z):
        {
            int threads_per_block(THREADS_PER_BLOCK);
            int num_blocks((n_cnt / threads_per_block) + 1);
            k_derive_activation<<<num_blocks, threads_per_block>>>(n->get_arr(Gpu),
                                                                   agg->get_arr(Gpu), n_cnt, activation_func_id);
        }

        // Derive z (weight):
        {
            int threads_per_block(THREADS_PER_BLOCK);
            int num_blocks(((n_cnt * prv_n_cnt) / threads_per_block) + 1);
            k_derive_z_and_increment_weight_derivative<<<num_blocks, threads_per_block>>>(agg->get_arr(Gpu),
                                                                                          prv_n->get_arr(Gpu),
                                                                                          prv_dw->get_arr(Gpu),
                                                                                          n_cnt, prv_n_cnt);
        }

        // Derive z (bias):
        {
            int threads_per_block(THREADS_PER_BLOCK);
            int num_blocks((n_cnt / threads_per_block) + 1);
            k_derive_z_and_increment_bias_derivative<<<num_blocks, threads_per_block>>>(agg->get_arr(Gpu), prv_db->get_arr(Gpu), n_cnt);
        }

        // Derive z (activation) and aggregate derivatives:
        {
            if (lyr_idx > 1)
            {
                Tensor *temp_agg = new Tensor(1, prv_n_cnt, Gpu);
                temp_agg->set_all(0.0f);

                {
                    int threads_per_block(THREADS_PER_BLOCK);
                    int num_blocks(((prv_n_cnt * n_cnt) / threads_per_block) + 1);
                    k_derive_z_and_aggregate_derivatives<<<num_blocks, threads_per_block>>>(prv_w->get_arr(Gpu),
                                                                                            agg->get_arr(Gpu), temp_agg->get_arr(Gpu),
                                                                                            prv_n_cnt, n_cnt);
                }

                delete agg;
                agg = temp_agg;
            }
        }
    }

    delete agg;
}

void NN::optimize(int batch_size)
{

    int lyr_cnt = this->neurons.size();
    int lst_lyr_idx = lyr_cnt - 1;

    for (int lyr_idx = 0; lyr_idx < lst_lyr_idx; lyr_idx++)
    {
        Tensor *w = this->weights[lyr_idx];
        Tensor *b = this->biases[lyr_idx];
        Tensor *dw = this->weight_derivatives[lyr_idx];
        Tensor *db = this->bias_derivatives[lyr_idx];

        int n_cnt = w->get_col_cnt();
        int nxt_n_cnt = w->get_row_cnt();

        // Weights:
        {
            int threads_per_block(THREADS_PER_BLOCK);
            int num_blocks(((nxt_n_cnt * n_cnt) / threads_per_block) + 1);
            k_adjust_weight<<<num_blocks, threads_per_block>>>(w->get_arr(Gpu), dw->get_arr(Gpu), batch_size, this->learning_rate,
                                                               (nxt_n_cnt * n_cnt));
        }

        // Biases:
        {
            int threads_per_block(THREADS_PER_BLOCK);
            int num_blocks((nxt_n_cnt / threads_per_block) + 1);
            k_adjust_bias<<<num_blocks, threads_per_block>>>(b->get_arr(Gpu), db->get_arr(Gpu), batch_size, this->learning_rate, nxt_n_cnt);
        }
    }
}

void NN::check_gradient(Tensor *x, Tensor *y, bool print_flg)
{
    float agg_ana_grad = 0.0f;
    float agg_num_grad = 0.0f;
    float agg_grad_diff = 0.0f;

    float epsilon = 0.001f;

    // Analytical gradients:
    {
        this->feed_forward(x);
        this->back_propagate(y);
    }

    // Numerical gradients:
    {
        int lyr_cnt = this->neurons.size();
        int lst_lyr_idx = lyr_cnt - 1;

        for (int lyr_idx = 0; lyr_idx < lst_lyr_idx; lyr_idx++)
        {
            Tensor *w = this->weights[lyr_idx];
            Tensor *b = this->biases[lyr_idx];
            Tensor *dw = this->weight_derivatives[lyr_idx];
            Tensor *db = this->bias_derivatives[lyr_idx];

            // Weights:
            for (int w_idx = 0; w_idx < w->get_row_cnt() * w->get_col_cnt(); w_idx++)
            {
                float left_cost = 0.0;
                float right_cost = 0.0;

                float orig_w_val = w->get_idx(w_idx);

                float left_w_val = orig_w_val - epsilon;
                float right_w_val = orig_w_val + epsilon;

                float ana_grad = dw->get_idx(w_idx);

                // Left:
                w->set_idx(w_idx, left_w_val);
                {
                    this->feed_forward(x);
                    left_cost += this->get_cost(y);
                }

                // Right:
                w->set_idx(w_idx, right_w_val);
                {
                    this->feed_forward(x);
                    right_cost += this->get_cost(y);
                }

                float num_grad = (right_cost - left_cost) / (2.0f * epsilon);

                if (print_flg)
                {
                    printf("W: %d  %d\t%f : %f  (%f)\n", lyr_idx, w_idx, ana_grad, num_grad, fabs(ana_grad - num_grad));
                }

                agg_ana_grad += pow(ana_grad, 2.0f);
                agg_num_grad += pow(num_grad, 2.0f);
                agg_grad_diff += pow(ana_grad - num_grad, 2.0f);

                w->set_idx(w_idx, orig_w_val);
            }

            // Biases:
            for (int b_idx = 0; b_idx < b->get_row_cnt(); b_idx++)
            {
                float left_cost = 0.0;
                float right_cost = 0.0;

                float orig_b_val = b->get_idx(b_idx);

                float left_b_val = orig_b_val - epsilon;
                float right_b_val = orig_b_val + epsilon;

                float ana_grad = db->get_idx(b_idx);

                // Left:
                b->set_idx(b_idx, left_b_val);
                {
                    this->feed_forward(x);
                    left_cost += this->get_cost(y);
                }

                // Right:
                b->set_idx(b_idx, right_b_val);
                {
                    this->feed_forward(x);
                    right_cost += this->get_cost(y);
                }

                float num_grad = (right_cost - left_cost) / (2.0f * epsilon);

                if (print_flg)
                {
                    printf("B: %d  %d\t%f : %f  (%f)\n", lyr_idx, b_idx, ana_grad, num_grad, fabs(ana_grad - num_grad));
                }

                agg_ana_grad += pow(ana_grad, 2.0f);
                agg_num_grad += pow(num_grad, 2.0f);
                agg_grad_diff += pow(ana_grad - num_grad, 2.0f);

                b->set_idx(b_idx, orig_b_val);
            }
        }
    }

    if ((agg_grad_diff) == 0.0f && (agg_ana_grad + agg_num_grad) == 0.0f)
    {
        printf("GRADIENT CHECK RESULT: %f\n", 0.0f);
    }
    else
    {
        printf("GRADIENT CHECK RESULT: %f\n", (agg_grad_diff) / (agg_ana_grad + agg_num_grad));
    }
}

void NN::profile(Tensor *x, Tensor *y)
{
    int epoch_cnt = 100;
    int batch_size = 100;

    printf("START PERFORMANCE TEST\n");
    clock_t t;
    t = clock();

    for (int epoch = 0; epoch < epoch_cnt; epoch++)
    {
        for (int batch = 0; batch < batch_size; batch++)
        {
            this->feed_forward(x);
            //this->get_cost(y);
            this->back_propagate(y);
        }
        //this->optimize(batch_size);
    }

    t = clock() - t;
    double time_taken = ((double)t) / CLOCKS_PER_SEC;

    printf("END PERFORMANCE TEST\n");
    printf("Elapsed Seconds: %f\n\n", time_taken);
}

ProgressReport NN::train(Batch *batch)
{
    ProgressReport rpt;

    int batch_size = batch->get_size();

    rpt.crct_cnt = 0;
    rpt.tot_cnt = batch_size;

    float cost = 0.0f;

    int lst_lyr_idx = this->neurons.size() - 1;

    for (int i = 0; i < batch_size; i++)
    {
        Tensor *x = batch->get_x(i);
        Tensor *y = batch->get_y(i);

        this->feed_forward(x);
        cost += this->get_cost(y);
        this->back_propagate(y);

        TensorTuple max_tup = this->neurons[lst_lyr_idx]->get_max();
        if (y->get_idx(max_tup.idx) == 1.0f)
        {
            rpt.crct_cnt++;
        }
    }

    cost /= batch_size;

    rpt.cost = cost;

    this->optimize(batch_size);

    return rpt;
}

ProgressReport NN::validate(Batch *batch)
{
    ProgressReport rpt;

    int batch_size = batch->get_size();

    float cost = 0.0f;

    for (int i = 0; i < batch_size; i++)
    {
        Tensor *x = batch->get_x(i);
        Tensor *y = batch->get_y(i);

        this->feed_forward(x);
        cost += this->get_cost(y);
    }

    cost /= batch_size;

    rpt.cost = cost;

    return rpt;
}

ProgressReport NN::test(Batch *batch)
{
    ProgressReport rpt;

    int batch_size = batch->get_size();

    float cost = 0.0f;

    for (int i = 0; i < batch_size; i++)
    {
        Tensor *x = batch->get_x(i);
        Tensor *y = batch->get_y(i);

        this->feed_forward(x);
        cost += this->get_cost(y);
    }

    cost /= batch_size;

    rpt.cost = cost;

    return rpt;
}

void NN::all(Supervisor *supervisor, int train_batch_size, int validation_chk_freq, const char *train_csv_path, const char *validation_csv_path)
{
    FILE *train_csv_file_ptr = fopen(train_csv_path, "w");
    FILE *validation_csv_file_ptr = fopen(validation_csv_path, "w");

    NN::write_csv_header(train_csv_file_ptr);
    NN::write_csv_header(validation_csv_file_ptr);

    Batch *validation_batch = supervisor->create_validation_batch();
    float prv_validation_cost = FLT_MAX;

    Batch *test_batch = supervisor->create_test_batch();

    unsigned long int epoch = 1;
    while (true)
    {
        Batch *train_batch = supervisor->create_train_batch(train_batch_size);
        ProgressReport train_rpt = this->train(train_batch);
        NN::write_to_csv(train_csv_file_ptr, epoch, train_rpt);
        delete train_batch;

        if (epoch % validation_chk_freq == 0)
        {
            ProgressReport validation_rpt = this->validate(validation_batch);
            NN::write_to_csv(validation_csv_file_ptr, epoch, validation_rpt);

            if (prv_validation_cost <= validation_rpt.cost)
            {
                break;
            }

            prv_validation_cost = validation_rpt.cost;
        }

        epoch++;
    }

    ProgressReport test_rpt = this->test(test_batch);

    delete validation_batch;
    delete test_batch;

    fclose(train_csv_file_ptr);
    fclose(validation_csv_file_ptr);
}