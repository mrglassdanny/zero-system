#include "hip/hip_runtime.h"
#include "NN.cuh"

#define THREADS_PER_BLOCK 16

__device__ float d_relu(float z)
{
    return z > 0.0f ? z : 0.0f;
}

__device__ float d_derive_relu(float a)
{
    return a > 0.0f ? 1.0f : 0.0f;
}

__device__ float d_mse_cost(float p, float y)
{
    return pow((p - y), 2.0);
}

__device__ float d_derive_mse_cost(float p, float y)
{
    return 2.0f * (p - y);
}

__global__ void k_set_arr(float *arr, int cnt, float val)
{
    int tid = blockIdx.x * blockDim.x + threadIdx.x;

    if (tid < cnt)
    {
        arr[tid] = val;
    }
}

__global__ void k_dot_all(float *neu_arr, float *wgt_arr, float *nxt_neu_arr, int neu_cnt, int nxt_neu_cnt)
{
    __shared__ float temp[THREADS_PER_BLOCK];
    memset(temp, 0, THREADS_PER_BLOCK * sizeof(float));

    int tid = blockIdx.x * blockDim.x + threadIdx.x;

    int tot_cnt = neu_cnt * nxt_neu_cnt;

    if (tid < tot_cnt)
    {
        temp[threadIdx.x] = neu_arr[tid % neu_cnt] * wgt_arr[tid];
    }

    __syncthreads();

    if (threadIdx.x == 0)
    {
        // NOTE: this only works if we assume the threadIdx.x is 0!!!
        int a = tid / neu_cnt;
        int b = (tid + THREADS_PER_BLOCK - 1) / neu_cnt;

        if (neu_cnt >= THREADS_PER_BLOCK)
        {
            if (a == b)
            {
                float sum = 0.0f;

                for (int i = 0; i < THREADS_PER_BLOCK; i++)
                {
                    sum += temp[i];
                }
                atomicAdd(&nxt_neu_arr[tid / neu_cnt], sum);
            }
            else
            {
                float sums[2] = {0.0f, 0.0f};

                for (int i = 0; i < THREADS_PER_BLOCK; i++)
                {
                    if ((tid + i) / neu_cnt == a)
                    {
                        sums[0] += temp[i];
                    }
                    else
                    {
                        sums[1] += temp[i];
                    }
                }

                atomicAdd(&nxt_neu_arr[a], sums[0]);
                if (a + 1 < neu_cnt)
                {
                    atomicAdd(&nxt_neu_arr[a + 1], sums[1]);
                }
            }
        }
        else
        {
            for (int i = 0; i < THREADS_PER_BLOCK; i++)
            {
                atomicAdd(&nxt_neu_arr[(tid + i) / neu_cnt], temp[i]);
            }
        }
    }
}

__global__ void k_add_bias(float *bia_arr, float *nxt_neu_arr, int nxt_neu_cnt)
{
    int tid = blockIdx.x * blockDim.x + threadIdx.x;

    if (tid < nxt_neu_cnt)
    {
        nxt_neu_arr[tid] += bia_arr[tid];
    }
}

__global__ void k_activate(float *neu_arr, int neu_cnt, ActivationFunctionId activation_func_id)
{
    int tid = blockIdx.x * blockDim.x + threadIdx.x;

    if (tid < neu_cnt)
    {
        switch (activation_func_id)
        {
        case ReLU:
            neu_arr[tid] = d_relu(neu_arr[tid]);
            break;
        case Sigmoid:
            neu_arr[tid] = d_relu(neu_arr[tid]);
            break;
        case Tanh:
            neu_arr[tid] = d_relu(neu_arr[tid]);
            break;
        default:
            // None
            break;
        }
    }
}

__global__ void k_cost(float *neu_arr, float *y_arr, float *atomic_cost, int neu_cnt)
{
    __shared__ float temp[THREADS_PER_BLOCK];
    memset(temp, 0, THREADS_PER_BLOCK * sizeof(float));

    int tid = blockIdx.x * blockDim.x + threadIdx.x;

    if (tid < neu_cnt)
    {
        temp[threadIdx.x] = d_mse_cost(neu_arr[tid], y_arr[tid]);
    }

    __syncthreads();

    if (threadIdx.x == 0)
    {
        float cost = 0.0f;
        for (int i = 0; i < THREADS_PER_BLOCK; i++)
        {
            cost += temp[i];
        }

        atomicAdd(atomic_cost, cost);
    }
}

__global__ void k_derive_cost(float *neu_arr, float *y_arr, float *agg_arr, int neu_cnt, CostFunctionId cost_func_id)
{
    int tid = blockIdx.x * blockDim.x + threadIdx.x;

    if (tid < neu_cnt)
    {
        switch (cost_func_id)
        {
        case MSE:
            agg_arr[tid] *= d_derive_mse_cost(neu_arr[tid], y_arr[tid]);
            break;
        case CrossEntropy:
            agg_arr[tid] *= d_derive_mse_cost(neu_arr[tid], y_arr[tid]);
            break;
        default:
            break;
        }
    }
}

__global__ void k_derive_activation(float *neu_arr, float *agg_arr, int neu_cnt, ActivationFunctionId activation_func_id)
{
    int tid = blockIdx.x * blockDim.x + threadIdx.x;

    if (tid < neu_cnt)
    {
        switch (activation_func_id)
        {
        case ReLU:
            agg_arr[tid] *= d_derive_relu(neu_arr[tid]);
            break;
        case Sigmoid:
            agg_arr[tid] *= d_derive_relu(neu_arr[tid]);
            break;
        case Tanh:
            agg_arr[tid] *= d_derive_relu(neu_arr[tid]);
            break;
        default:
            // None
            break;
        }
    }
}

__global__ void k_derive_n_increment_weights(float *agg_arr, float *neu_arr, float *dw_arr, int nxt_neu_cnt, int neu_cnt)
{
    int tid = blockIdx.x * blockDim.x + threadIdx.x;

    int nxt_neu_idx = tid / neu_cnt;
    int neu_idx = tid % neu_cnt;
    int wgt_idx = nxt_neu_idx * neu_cnt + neu_idx;

    if (tid < (nxt_neu_cnt * neu_cnt))
    {
        dw_arr[wgt_idx] += (agg_arr[nxt_neu_idx] * neu_arr[neu_idx]);
    }
}

__global__ void k_derive_n_increment_biases(float *agg_arr, float *db_arr, int cnt)
{
    int tid = blockIdx.x * blockDim.x + threadIdx.x;

    if (tid < cnt)
    {
        db_arr[tid] += (agg_arr[tid]);
    }
}

__global__ void k_aggregate_derivatives(float *wgt_arr, float *agg_arr, float *out_arr, int col_cnt, int row_cnt)
{
    __shared__ float temp[THREADS_PER_BLOCK];
    memset(temp, 0, THREADS_PER_BLOCK * sizeof(float));

    int tid = blockIdx.x * blockDim.x + threadIdx.x;

    int tot_cnt = col_cnt * row_cnt;

    int row_idx = tid / col_cnt;
    int col_idx = tid % col_cnt;
    int wgt_idx = row_idx * col_cnt + col_idx;

    if (tid < tot_cnt)
    {
        temp[threadIdx.x] = (agg_arr[row_idx] * wgt_arr[wgt_idx]);
    }

    __syncthreads();

    if (threadIdx.x == 0)
    {
        for (int i = 0; i < THREADS_PER_BLOCK; i++)
        {
            // NOTE: this only works if we assume the threadIdx.x is 0!!!
            int idx = (tid + i) % col_cnt;
            atomicAdd(&out_arr[idx], temp[i]);
        }
    }
}

__global__ void k_optimize_weights(float *wgt_arr, float *dw_arr, int batch_size, float lr, int cnt)
{
    int tid = blockIdx.x * blockDim.x + threadIdx.x;

    if (tid < cnt)
    {
        wgt_arr[tid] -= ((dw_arr[tid] * lr) / (float)batch_size);
        dw_arr[tid] = 0.0f;
    }
}

__global__ void k_optimize_biases(float *bia_arr, float *db_arr, int batch_size, float lr, int cnt)
{
    int tid = blockIdx.x * blockDim.x + threadIdx.x;

    if (tid < cnt)
    {
        bia_arr[tid] -= ((db_arr[tid] * lr) / (float)batch_size);
        db_arr[tid] = 0.0f;
    }
}

NN::NN(std::vector<int> layer_config, ActivationFunctionId hidden_layer_activation_func_id,
       ActivationFunctionId output_layer_activation_func_id, CostFunctionId cost_func_id, float learning_rate)
{
    // Leave input neurons NULL for now!
    this->neurons.push_back(nullptr);
    for (int lyr_idx = 0; lyr_idx < layer_config.size() - 1; lyr_idx++)
    {
        this->neurons.push_back(new Tensor(1, layer_config[lyr_idx + 1], Gpu));
        this->weights.push_back(new Tensor(layer_config[lyr_idx + 1], layer_config[lyr_idx], Gpu));
        this->biases.push_back(new Tensor(layer_config[lyr_idx + 1], 1, Gpu));
        this->weight_derivatives.push_back(new Tensor(layer_config[lyr_idx + 1], layer_config[lyr_idx], Gpu));
        this->bias_derivatives.push_back(new Tensor(layer_config[lyr_idx + 1], 1, Gpu));
    }

    this->hidden_layer_activation_func_id = hidden_layer_activation_func_id;
    this->output_layer_activation_func_id = output_layer_activation_func_id;

    this->cost_func_id = cost_func_id;

    this->learning_rate = learning_rate;
}

NN::~NN()
{
}

void NN::feed_forward(Tensor *x)
{
    x->translate(Gpu);
    this->neurons[0] = x;

    int lst_lyr_idx = this->neurons.size() - 1;

    for (int lyr_idx = 0; lyr_idx < lst_lyr_idx; lyr_idx++)
    {
        int neu_cnt = this->weights[lyr_idx]->get_col_cnt();
        int nxt_neu_cnt = this->weights[lyr_idx]->get_row_cnt();

        Tensor *neu = this->neurons[lyr_idx];
        Tensor *wgt = this->weights[lyr_idx];
        Tensor *bia = this->biases[lyr_idx];
        Tensor *nxt_neu = this->neurons[lyr_idx + 1];

        ActivationFunctionId activation_func_id = (lyr_idx == lst_lyr_idx - 1) ? this->output_layer_activation_func_id : this->hidden_layer_activation_func_id;

        // Need to reset neurons before we do anything:
        {
            int threads_per_block(THREADS_PER_BLOCK);
            int num_blocks(ceil((float)nxt_neu_cnt / (float)threads_per_block));
            k_set_arr<<<num_blocks, threads_per_block>>>(nxt_neu->get_arr(Gpu), nxt_neu_cnt, 0.0f);
        }

        // Dot product:
        {
            int threads_per_block(THREADS_PER_BLOCK);
            int num_blocks(ceil((float)(neu_cnt * nxt_neu_cnt) / (float)threads_per_block));
            k_dot_all<<<num_blocks, threads_per_block>>>(neu->get_arr(Gpu), wgt->get_arr(Gpu),
                                                         nxt_neu->get_arr(Gpu), neu_cnt, nxt_neu_cnt);
        }

        // Add biases:
        {
            int threads_per_block(THREADS_PER_BLOCK);
            int num_blocks(ceil((float)nxt_neu_cnt / (float)threads_per_block));
            k_add_bias<<<num_blocks, threads_per_block>>>(bia->get_arr(Gpu), nxt_neu->get_arr(Gpu),
                                                          nxt_neu_cnt);
        }

        // Activate:
        {
            int threads_per_block(THREADS_PER_BLOCK);
            int num_blocks(ceil((float)nxt_neu_cnt / (float)threads_per_block));
            k_activate<<<num_blocks, threads_per_block>>>(nxt_neu->get_arr(Gpu), nxt_neu_cnt, activation_func_id);
        }
    }
}

float NN::get_cost(Tensor *y, int batch_size)
{
    float h_cost = 0.0f;
    float *d_cost;
    hipMalloc(&d_cost, sizeof(float));

    int lyr_cnt = this->neurons.size();
    int lst_lyr_idx = lyr_cnt - 1;
    int lst_lyr_neu_cnt = this->neurons[lst_lyr_idx]->get_col_cnt();

    Tensor *lst_lyr_neu = this->neurons[lst_lyr_idx];

    {
        int threads_per_block(THREADS_PER_BLOCK);
        int num_blocks(ceil((float)lst_lyr_neu_cnt / (float)threads_per_block));

        // Aggregate async:
        k_cost<<<num_blocks, threads_per_block>>>(lst_lyr_neu->get_arr(Gpu), y->get_arr(Gpu),
                                                  d_cost, lst_lyr_neu_cnt);
    }

    hipMemcpy(&h_cost, d_cost, sizeof(float), hipMemcpyDeviceToHost);
    hipFree(d_cost);

    // Divide sync:
    h_cost /= (float)batch_size;

    return h_cost;
}

void NN::back_propagate(Tensor *y)
{

    int lyr_cnt = this->neurons.size();
    int lst_lyr_idx = lyr_cnt - 1;
    int lst_lyr_neu_cnt = this->neurons[lst_lyr_idx]->get_col_cnt();

    Tensor *agg = new Tensor(1, lst_lyr_neu_cnt, Gpu);
    agg->set_all(1.0f);

    // Cost:
    {
        int threads_per_block(THREADS_PER_BLOCK);
        int num_blocks(ceil((float)lst_lyr_neu_cnt / (float)threads_per_block));
        k_derive_cost<<<num_blocks, threads_per_block>>>(this->neurons[lst_lyr_idx]->get_arr(Gpu),
                                                         y->get_arr(Gpu), agg->get_arr(Gpu), lst_lyr_neu_cnt, this->cost_func_id);
    }

    for (int lyr_idx = lst_lyr_idx; lyr_idx > 0; lyr_idx--)
    {
        int neu_cnt = this->weights[lyr_idx]->get_row_cnt();
        int prv_neu_cnt = this->weights[lyr_idx]->get_col_cnt();

        Tensor *neu = this->neurons[lyr_idx];
        Tensor *prv_neu = this->neurons[lyr_idx - 1];
        Tensor *prv_wgt = this->weights[lyr_idx - 1];
        Tensor *prv_bia = this->biases[lyr_idx - 1];
        Tensor *prv_dw = this->weight_derivatives[lyr_idx - 1];
        Tensor *prv_db = this->bias_derivatives[lyr_idx - 1];

        ActivationFunctionId activation_func_id = (lyr_idx == lst_lyr_idx - 1) ? this->output_layer_activation_func_id : this->hidden_layer_activation_func_id;

        // Activations:
        {
            int threads_per_block(THREADS_PER_BLOCK);
            int num_blocks(ceil((float)neu_cnt / (float)threads_per_block));
            k_derive_activation<<<num_blocks, threads_per_block>>>(neu->get_arr(Gpu),
                                                                   agg->get_arr(Gpu), neu_cnt, activation_func_id);
        }

        // Weights:
        {
            int threads_per_block(THREADS_PER_BLOCK);
            int num_blocks(ceil((float)neu_cnt * prv_neu_cnt / (float)threads_per_block));
            k_derive_n_increment_weights<<<num_blocks, threads_per_block>>>(agg->get_arr(Gpu),
                                                                            prv_neu->get_arr(Gpu),
                                                                            prv_dw->get_arr(Gpu),
                                                                            neu_cnt, prv_neu_cnt);
        }

        // Biases:
        {
            int threads_per_block(THREADS_PER_BLOCK);
            int num_blocks(ceil((float)neu_cnt / (float)threads_per_block));
            k_derive_n_increment_biases<<<num_blocks, threads_per_block>>>(agg->get_arr(Gpu), prv_db->get_arr(Gpu), neu_cnt);
        }

        // Aggregate:
        {
            if (lyr_idx > 1)
            {
                Tensor *temp_agg = new Tensor(1, prv_neu_cnt, Gpu);
                temp_agg->set_all(0.0f);

                {
                    int threads_per_block(THREADS_PER_BLOCK);
                    int num_blocks(ceil((float)prv_neu_cnt * neu_cnt / (float)threads_per_block));
                    k_aggregate_derivatives<<<num_blocks, threads_per_block>>>(prv_wgt->get_arr(Gpu),
                                                                               agg->get_arr(Gpu), temp_agg->get_arr(Gpu),
                                                                               prv_neu_cnt, neu_cnt);
                }

                delete agg;
                agg = new Tensor(temp_agg);
                delete temp_agg;
            }
        }
    }

    delete agg;
}

void NN::optimize(int batch_size)
{

    int lyr_cnt = this->neurons.size();
    int lst_lyr_idx = lyr_cnt - 1;

    for (int lyr_idx = 0; lyr_idx < lst_lyr_idx; lyr_idx++)
    {
        int neu_cnt = this->weights[lyr_idx]->get_col_cnt();
        int nxt_neu_cnt = this->weights[lyr_idx]->get_row_cnt();

        Tensor *wgt = this->weights[lyr_idx];
        Tensor *bia = this->biases[lyr_idx];
        Tensor *dw = this->weight_derivatives[lyr_idx];
        Tensor *db = this->bias_derivatives[lyr_idx];

        // Weights:
        {
            int threads_per_block(THREADS_PER_BLOCK);
            int num_blocks(ceil(((float)nxt_neu_cnt * neu_cnt) / (float)threads_per_block));
            k_optimize_weights<<<num_blocks, threads_per_block>>>(wgt->get_arr(Gpu), dw->get_arr(Gpu), batch_size, this->learning_rate,
                                                                  (nxt_neu_cnt * neu_cnt));
        }

        // Biases:
        {
            int threads_per_block(THREADS_PER_BLOCK);
            int num_blocks(ceil((float)nxt_neu_cnt / (float)threads_per_block));
            k_optimize_biases<<<num_blocks, threads_per_block>>>(bia->get_arr(Gpu), db->get_arr(Gpu), batch_size, this->learning_rate, nxt_neu_cnt);
        }
    }
}