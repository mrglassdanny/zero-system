#include "hip/hip_runtime.h"
#include "layer.cuh"

using namespace zero_v2::core;
using namespace zero_v2::nn;

// Device functions:

__device__ float d_relu(float val)
{
    return val > 0.0f ? val : 0.0f;
}

__device__ float d_derive_relu(float val)
{
    return val > 0.0f ? 1.0f : 0.0f;
}

__device__ float d_sigmoid(float val)
{
    return (1.0 / (1.0 + exp(-val)));
}

__device__ float d_derive_sigmoid(float val)
{
    return (val) * (1.0 - val);
}

__device__ float d_tanh(float val)
{
    return ((exp(val) - exp(-val)) / (exp(val) + exp(-val)));
}

__device__ float d_derive_tanh(float val)
{
    return (1 - (val * val));
}

__device__ float d_sine(float val)
{
    return sin(val);
}

__device__ float d_derive_sine(float val)
{
    return cos(val);
}

__device__ float d_cosine(float val)
{
    return cos(val);
}

__device__ float d_derive_cosine(float val)
{
    return -sin(val);
}

// Kernel functions:

__global__ void k_dot(float *n_arr, float *w_arr, float *nxt_n_arr, int n_cnt, int nxt_n_cnt)
{
    __shared__ float temp[CUDA_THREADS_PER_BLOCK];
    memset(temp, 0, CUDA_THREADS_PER_BLOCK * sizeof(float));

    int tid = blockIdx.x * blockDim.x + threadIdx.x;

    int w_cnt = n_cnt * nxt_n_cnt;

    int n_idx = tid % n_cnt;
    int w_idx = tid;

    if (w_idx < w_cnt)
    {
        temp[threadIdx.x] = n_arr[n_idx] * w_arr[w_idx];
    }

    __syncthreads();

    if (threadIdx.x == 0) // threadIdx MUST be 0 for below logic to work!
    {
        /*
        The goal here is to try to minimize atomic adds. If the neuron count is
        greater than or equal to the threads per block, a maximum of 2 atomic adds
        is necessary for this block. However, most of the time we can get away with just 1.

        If the threads per block is greater than the neuron count, we just play it safe
        and incur an atomic add for each thread in the block.
        */

        int lower_idx = tid / n_cnt;
        int upper_idx = ((tid + CUDA_THREADS_PER_BLOCK) - 1) / n_cnt;

        if (n_cnt >= CUDA_THREADS_PER_BLOCK)
        {
            if (lower_idx == upper_idx)
            {
                float sum = 0.0f;

#pragma unroll
                for (int i = 0; i < CUDA_THREADS_PER_BLOCK; i++)
                {
                    sum += temp[i];
                }

                atomicAdd(&nxt_n_arr[lower_idx], sum);
            }
            else
            {
                float sums[2] = {0.0f, 0.0f};

#pragma unroll
                for (int i = 0; i < CUDA_THREADS_PER_BLOCK; i++)
                {
                    if ((tid + i) / n_cnt == lower_idx)
                    {
                        sums[0] += temp[i];
                    }
                    else
                    {
                        sums[1] += temp[i];
                    }
                }

                atomicAdd(&nxt_n_arr[lower_idx], sums[0]);
                if (upper_idx < nxt_n_cnt)
                {
                    atomicAdd(&nxt_n_arr[upper_idx], sums[1]);
                }
            }
        }
        else
        {

#pragma unroll
            for (int i = 0; i < CUDA_THREADS_PER_BLOCK; i++)
            {
                atomicAdd(&nxt_n_arr[(tid + i) / n_cnt], temp[i]);
            }
        }
    }
}

__global__ void k_add_bias(float *b_arr, float *nxt_n_arr, int nxt_n_cnt)
{
    int tid = blockIdx.x * blockDim.x + threadIdx.x;

    if (tid < nxt_n_cnt)
    {
        nxt_n_arr[tid] += b_arr[tid];
    }
}

__global__ void k_derive_z_and_increment_weight_derivative(float *dc_arr, float *n_arr, float *dw_arr, int dc_cnt, int n_cnt)
{
    int tid = blockIdx.x * blockDim.x + threadIdx.x;

    int w_cnt = dc_cnt * n_cnt;

    int nxt_n_idx = tid % n_cnt;
    int n_idx = tid / n_cnt;
    int w_idx = n_idx * n_cnt + nxt_n_idx;

    if (w_idx < w_cnt)
    {
        dw_arr[w_idx] += (dc_arr[n_idx] * n_arr[nxt_n_idx]);
    }
}

__global__ void k_derive_z_and_increment_bias_derivative(float *dc_arr, float *db_arr, int dc_cnt)
{
    int tid = blockIdx.x * blockDim.x + threadIdx.x;

    if (tid < dc_cnt)
    {
        db_arr[tid] += (dc_arr[tid]);
    }
}

__global__ void k_activate(float *n_arr, float *nxt_n_arr, int n_cnt, ActivationFunction typ)
{
    int tid = blockIdx.x * blockDim.x + threadIdx.x;

    if (tid < n_cnt)
    {
        switch (typ)
        {
        case ActivationFunction::ReLU:
            nxt_n_arr[tid] = d_relu(n_arr[tid]);
            break;
        case ActivationFunction::Sigmoid:
            nxt_n_arr[tid] = d_sigmoid(n_arr[tid]);
            break;
        case ActivationFunction::Tanh:
            nxt_n_arr[tid] = d_tanh(n_arr[tid]);
            break;
        case ActivationFunction::Sine:
            nxt_n_arr[tid] = d_sine(n_arr[tid]);
            break;
        case ActivationFunction::Cosine:
            nxt_n_arr[tid] = d_cosine(n_arr[tid]);
            break;
        default:
            // None
            break;
        }
    }
}

__global__ void k_derive_activation(float *n_arr, float *dc_arr, int n_cnt, ActivationFunction typ)
{
    int tid = blockIdx.x * blockDim.x + threadIdx.x;

    if (tid < n_cnt)
    {
        switch (typ)
        {
        case ActivationFunction::ReLU:
            dc_arr[tid] *= d_derive_relu(n_arr[tid]);
            break;
        case ActivationFunction::Sigmoid:
            dc_arr[tid] *= d_derive_sigmoid(n_arr[tid]);
            break;
        case ActivationFunction::Tanh:
            dc_arr[tid] *= d_derive_tanh(n_arr[tid]);
            break;
        case ActivationFunction::Sine:
            dc_arr[tid] *= d_derive_sine(n_arr[tid]);
            break;
        case ActivationFunction::Cosine:
            dc_arr[tid] *= d_derive_cosine(n_arr[tid]);
            break;
        default:
            // None
            break;
        }
    }
}

// Layer functions:

Layer::Layer()
{
    this->n = nullptr;
}

Layer::~Layer()
{
    if (this->n != nullptr)
    {
        delete this->n;
    }
}

LinearLayer::LinearLayer(int n_cnt, int nxt_n_cnt, WeightInitializationType wgt_init_typ)
    : Layer()
{
    this->n = new Tensor(Device::Cuda, n_cnt);
    this->n->reset();

    this->w = new Tensor(Device::Cuda, nxt_n_cnt, n_cnt);
    this->b = new Tensor(Device::Cuda, nxt_n_cnt);
    switch (wgt_init_typ)
    {
    case WeightInitializationType::He:
        this->w->set_all_rand(0.0f, sqrt(2.0f / n_cnt));
        this->b->set_all_rand(0.0f, sqrt(2.0f / n_cnt));
        break;
    case WeightInitializationType::Xavier:
        this->w->set_all_rand(0.0f, sqrt(1.0f / n_cnt));
        this->b->set_all_rand(0.0f, sqrt(1.0f / n_cnt));
        break;
    case WeightInitializationType::Zeros:
        this->w->reset();
        this->b->reset();
        break;
    default:
        this->w->set_all_rand(0.0f, 1.0f);
        this->b->set_all_rand(0.0f, 1.0f);
        break;
    }

    this->dw = new Tensor(Device::Cuda, n_cnt, nxt_n_cnt);
    this->dw->reset();

    this->db = new Tensor(Device::Cuda, nxt_n_cnt);
    this->db->reset();
}

LinearLayer::~LinearLayer()
{
    delete this->w;
    delete this->b;
    delete this->dw;
    delete this->db;
}

void LinearLayer::evaluate(Tensor *nxt_n)
{
    int n_cnt = this->n->get_cnt();
    int nxt_n_cnt = nxt_n->get_cnt();

    // Dot product:
    {
        int threads_per_block = CUDA_THREADS_PER_BLOCK;
        int num_blocks = ((n_cnt * nxt_n_cnt) / threads_per_block) + 1;
        k_dot<<<num_blocks, threads_per_block>>>(this->n->get_arr(), w->get_arr(),
                                                 nxt_n->get_arr(), n_cnt, nxt_n_cnt);
    }

    // Add biases:
    {
        int threads_per_block(CUDA_THREADS_PER_BLOCK);
        int num_blocks((nxt_n_cnt / threads_per_block) + 1);
        k_add_bias<<<num_blocks, threads_per_block>>>(this->b->get_arr(), nxt_n->get_arr(),
                                                      nxt_n_cnt);
    }
}

void LinearLayer::derive(Tensor *dc)
{
    int dc_cnt = dc->get_cnt();
    int n_cnt = this->n->get_cnt();

    // Weights:
    {
        int threads_per_block = CUDA_THREADS_PER_BLOCK;
        int num_blocks = ((dc_cnt * n_cnt) / threads_per_block) + 1;
        k_derive_z_and_increment_weight_derivative<<<num_blocks, threads_per_block>>>(dc->get_arr(),
                                                                                      this->n->get_arr(),
                                                                                      this->dw->get_arr(),
                                                                                      dc_cnt, n_cnt);
    }

    // Biases:
    {
        int threads_per_block = CUDA_THREADS_PER_BLOCK;
        int num_blocks = (dc_cnt / threads_per_block) + 1;
        k_derive_z_and_increment_bias_derivative<<<num_blocks, threads_per_block>>>(dc->get_arr(), this->db->get_arr(), n_cnt);
    }
}

ActivationLayer::ActivationLayer(int n_cnt, ActivationFunction activation_fn)
    : Layer()
{
    this->n = new Tensor(Device::Cuda, n_cnt);
    this->n->reset();

    this->activation_fn = activation_fn;
}

ActivationLayer::~ActivationLayer()
{
}

void ActivationLayer::evaluate(Tensor *nxt_n)
{
    {
        int threads_per_block = CUDA_THREADS_PER_BLOCK;
        int num_blocks = (this->n->get_cnt() / threads_per_block) + 1;
        k_activate<<<num_blocks, threads_per_block>>>(this->n->get_arr(), nxt_n->get_arr(), this->n->get_cnt(), this->activation_fn);
    }
}

void ActivationLayer::derive(Tensor *dc)
{
    {
        int threads_per_block = CUDA_THREADS_PER_BLOCK;
        int num_blocks = (this->n->get_cnt() / threads_per_block) + 1;
        k_derive_activation<<<num_blocks, threads_per_block>>>(this->n->get_arr(), dc->get_arr(), this->n->get_cnt(), this->activation_fn);
    }
}
