#include "hip/hip_runtime.h"
#include "layer.cuh"

using namespace zero_v2::core;
using namespace zero_v2::nn;

// Device functions:

__device__ float d_relu(float val)
{
    return val > 0.0f ? val : 0.0f;
}

__device__ float d_derive_relu(float val)
{
    return val > 0.0f ? 1.0f : 0.0f;
}

__device__ float d_sigmoid(float val)
{
    return (1.0 / (1.0 + exp(-val)));
}

__device__ float d_derive_sigmoid(float val)
{
    return (val) * (1.0 - val);
}

__device__ float d_tanh(float val)
{
    return ((exp(val) - exp(-val)) / (exp(val) + exp(-val)));
}

__device__ float d_derive_tanh(float val)
{
    return (1 - (val * val));
}

__device__ float d_sine(float val)
{
    return sin(val);
}

__device__ float d_derive_sine(float val)
{
    return cos(val);
}

__device__ float d_cosine(float val)
{
    return cos(val);
}

__device__ float d_derive_cosine(float val)
{
    return -sin(val);
}

// Kernel functions:

__global__ void k_dot(float *n_arr, float *w_arr, float *nxt_n_arr, int n_cnt, int nxt_n_cnt)
{
    __shared__ float temp[CUDA_THREADS_PER_BLOCK];
    memset(temp, 0, CUDA_THREADS_PER_BLOCK * sizeof(float));

    int tid = blockIdx.x * blockDim.x + threadIdx.x;

    int w_cnt = n_cnt * nxt_n_cnt;

    int n_idx = tid % n_cnt;
    int w_idx = tid;

    if (w_idx < w_cnt)
    {
        temp[threadIdx.x] = n_arr[n_idx] * w_arr[w_idx];
    }

    __syncthreads();

    if (threadIdx.x == 0) // threadIdx MUST be 0 for below logic to work!
    {
        /*
        The goal here is to try to minimize atomic adds. If the neuron count is
        greater than or equal to the threads per block, a maximum of 2 atomic adds
        is necessary for this block. However, most of the time we can get away with just 1.

        If the threads per block is greater than the neuron count, we just play it safe
        and incur an atomic add for each thread in the block.
        */

        int lower_idx = tid / n_cnt;
        int upper_idx = ((tid + CUDA_THREADS_PER_BLOCK) - 1) / n_cnt;

        if (n_cnt >= CUDA_THREADS_PER_BLOCK)
        {
            if (lower_idx == upper_idx)
            {
                float sum = 0.0f;

#pragma unroll
                for (int i = 0; i < CUDA_THREADS_PER_BLOCK; i++)
                {
                    sum += temp[i];
                }

                atomicAdd(&nxt_n_arr[lower_idx], sum);
            }
            else
            {
                float sums[2] = {0.0f, 0.0f};

#pragma unroll
                for (int i = 0; i < CUDA_THREADS_PER_BLOCK; i++)
                {
                    if ((tid + i) / n_cnt == lower_idx)
                    {
                        sums[0] += temp[i];
                    }
                    else
                    {
                        sums[1] += temp[i];
                    }
                }

                atomicAdd(&nxt_n_arr[lower_idx], sums[0]);
                if (upper_idx < nxt_n_cnt)
                {
                    atomicAdd(&nxt_n_arr[upper_idx], sums[1]);
                }
            }
        }
        else
        {

#pragma unroll
            for (int i = 0; i < CUDA_THREADS_PER_BLOCK; i++)
            {
                atomicAdd(&nxt_n_arr[(tid + i) / n_cnt], temp[i]);
            }
        }
    }
}

__global__ void k_add_bias(float *b_arr, float *nxt_n_arr, int nxt_n_cnt)
{
    int tid = blockIdx.x * blockDim.x + threadIdx.x;

    if (tid < nxt_n_cnt)
    {
        nxt_n_arr[tid] += b_arr[tid];
    }
}

__global__ void k_derive_z_and_increment_weight_derivative(float *dc_arr, float *n_arr, float *dw_arr, int dc_cnt, int n_cnt)
{
    int tid = blockIdx.x * blockDim.x + threadIdx.x;

    int w_cnt = dc_cnt * n_cnt;

    int nxt_n_idx = tid % n_cnt;
    int n_idx = tid / n_cnt;
    int w_idx = n_idx * n_cnt + nxt_n_idx;

    if (w_idx < w_cnt)
    {
        dw_arr[w_idx] += (dc_arr[n_idx] * n_arr[nxt_n_idx]);
    }
}

__global__ void k_derive_z_and_increment_bias_derivative(float *dc_arr, float *db_arr, int dc_cnt)
{
    int tid = blockIdx.x * blockDim.x + threadIdx.x;

    if (tid < dc_cnt)
    {
        db_arr[tid] += (dc_arr[tid]);
    }
}

__global__ void k_derive_z_and_aggregate_derivatives(float *dc_arr, float *nxt_w_arr, float *nxt_dc_arr, int n_cnt, int nxt_n_cnt)
{
    __shared__ float temp[CUDA_THREADS_PER_BLOCK];
    memset(temp, 0, CUDA_THREADS_PER_BLOCK * sizeof(float));

    int tid = blockIdx.x * blockDim.x + threadIdx.x;

    int w_cnt = nxt_n_cnt * n_cnt;

    // Transpose the weights "matrix".
    int n_idx = tid % n_cnt;
    int nxt_n_idx = tid / n_cnt;
    int w_idx = n_idx * nxt_n_cnt + nxt_n_idx;

    if (w_idx < w_cnt)
    {
        temp[threadIdx.x] = (dc_arr[n_idx] * nxt_w_arr[w_idx]);
    }

    __syncthreads();

    if (threadIdx.x == 0) // threadIdx MUST be 0 for below logic to work!
    {
        /*
        The goal here is to try to minimize atomic adds. If the neuron count is
        greater than or equal to the threads per block, a maximum of 2 atomic adds
        is necessary for this block. However, most of the time we can get away with just 1.

        If the threads per block is greater than the neuron count, we just play it safe
        and incur an atomic add for each thread in the block.
        */

        int lower_idx = tid / n_cnt;
        int upper_idx = ((tid + CUDA_THREADS_PER_BLOCK) - 1) / n_cnt;

        if (n_cnt >= CUDA_THREADS_PER_BLOCK)
        {
            if (lower_idx == upper_idx)
            {
                float sum = 0.0f;

#pragma unroll
                for (int i = 0; i < CUDA_THREADS_PER_BLOCK; i++)
                {
                    sum += temp[i];
                }
                atomicAdd(&nxt_dc_arr[lower_idx], sum);
            }
            else
            {
                float sums[2] = {0.0f, 0.0f};

#pragma unroll
                for (int i = 0; i < CUDA_THREADS_PER_BLOCK; i++)
                {
                    if ((tid + i) / n_cnt == lower_idx)
                    {
                        sums[0] += temp[i];
                    }
                    else
                    {
                        sums[1] += temp[i];
                    }
                }

                atomicAdd(&nxt_dc_arr[lower_idx], sums[0]);
                if (upper_idx < nxt_n_cnt)
                {
                    atomicAdd(&nxt_dc_arr[upper_idx], sums[1]);
                }
            }
        }
        else
        {

#pragma unroll
            for (int i = 0; i < CUDA_THREADS_PER_BLOCK; i++)
            {
                atomicAdd(&nxt_dc_arr[(tid + i) / n_cnt], temp[i]);
            }
        }
    }
}

__global__ void k_adjust_weight(float *w_arr, float *dw_arr, int batch_size, float learning_rate, int cnt)
{
    int tid = blockIdx.x * blockDim.x + threadIdx.x;

    if (tid < cnt)
    {
        w_arr[tid] -= ((dw_arr[tid] * learning_rate) / (float)batch_size);
        dw_arr[tid] = 0.0f;
    }
}

__global__ void k_adjust_bias(float *b_arr, float *db_arr, int batch_size, float learning_rate, int cnt)
{
    int tid = blockIdx.x * blockDim.x + threadIdx.x;

    if (tid < cnt)
    {
        b_arr[tid] -= ((db_arr[tid] * learning_rate) / (float)batch_size);
        db_arr[tid] = 0.0f;
    }
}

__global__ void k_activate(float *n_arr, float *nxt_n_arr, int n_cnt, ActivationFunction activation_fn)
{
    int tid = blockIdx.x * blockDim.x + threadIdx.x;

    if (tid < n_cnt)
    {
        switch (activation_fn)
        {
        case ActivationFunction::ReLU:
            nxt_n_arr[tid] = d_relu(n_arr[tid]);
            break;
        case ActivationFunction::Sigmoid:
            nxt_n_arr[tid] = d_sigmoid(n_arr[tid]);
            break;
        case ActivationFunction::Tanh:
            nxt_n_arr[tid] = d_tanh(n_arr[tid]);
            break;
        case ActivationFunction::Sine:
            nxt_n_arr[tid] = d_sine(n_arr[tid]);
            break;
        case ActivationFunction::Cosine:
            nxt_n_arr[tid] = d_cosine(n_arr[tid]);
            break;
        default:
            // None
            break;
        }
    }
}

__global__ void k_derive_activation(float *n_arr, float *dc_arr, int n_cnt, ActivationFunction activation_fn)
{
    int tid = blockIdx.x * blockDim.x + threadIdx.x;

    if (tid < n_cnt)
    {
        switch (activation_fn)
        {
        case ActivationFunction::ReLU:
            dc_arr[tid] *= d_derive_relu(n_arr[tid]);
            break;
        case ActivationFunction::Sigmoid:
            dc_arr[tid] *= d_derive_sigmoid(n_arr[tid]);
            break;
        case ActivationFunction::Tanh:
            dc_arr[tid] *= d_derive_tanh(n_arr[tid]);
            break;
        case ActivationFunction::Sine:
            dc_arr[tid] *= d_derive_sine(n_arr[tid]);
            break;
        case ActivationFunction::Cosine:
            dc_arr[tid] *= d_derive_cosine(n_arr[tid]);
            break;
        default:
            // None
            break;
        }
    }
}

__global__ void k_set_dropout_mask(float *dropout_mask_arr, int dropout_mask_cnt, float dropout_rate)
{
    int tid = blockIdx.x * blockDim.x + threadIdx.x;

    if (tid < dropout_mask_cnt)
    {
        hiprandState state;
        hiprand_init(clock64(), tid, 0, &state);

        if (hiprand_uniform(&state) < dropout_rate)
        {
            dropout_mask_arr[tid] = 0.0f;
        }
        else
        {
            dropout_mask_arr[tid] = 1.0f;
        }
    }
}

__global__ void k_dropout(float *n_arr, float *dropout_mask_arr, float *nxt_n_arr, int n_cnt, float dropout_rate)
{
    int tid = blockIdx.x * blockDim.x + threadIdx.x;

    if (tid < n_cnt)
    {
        nxt_n_arr[tid] = n_arr[tid] * dropout_mask_arr[tid];
        nxt_n_arr[tid] = n_arr[tid] * (1.0f / (1.0f - dropout_rate));
    }
}

__global__ void k_derive_dropout(float *dc_arr, float *dropout_mask_arr, int n_cnt, float dropout_rate)
{
    int tid = blockIdx.x * blockDim.x + threadIdx.x;

    if (tid < n_cnt)
    {
        dc_arr[tid] *= dropout_mask_arr[tid];
        dc_arr[tid] *= (1.0f / (1.0f - dropout_rate));
    }
}

// Layer functions:

Layer::Layer()
{
    this->n = nullptr;
}

Layer::~Layer()
{
    if (this->n != nullptr)
    {
        delete this->n;
    }
}

// LearnableLayer functions:

LearnableLayer::LearnableLayer()
{
    this->w = nullptr;
    this->b = nullptr;
    this->dw = nullptr;
    this->db = nullptr;
}

LearnableLayer::~LearnableLayer()
{
    if (this->w != nullptr)
    {
        delete this->w;
    }

    if (this->b != nullptr)
    {
        delete this->b;
    }

    if (this->dw != nullptr)
    {
        delete this->dw;
    }

    if (this->db != nullptr)
    {
        delete this->db;
    }
}

// LinearLayer functions:

LinearLayer::LinearLayer()
    : LearnableLayer() {}

LinearLayer::LinearLayer(int n_cnt, int nxt_n_cnt, InitializationFunction init_fn)
    : LearnableLayer()
{
    this->n = new Tensor(Device::Cuda, n_cnt);
    this->n->reset();

    this->w = new Tensor(Device::Cuda, nxt_n_cnt, n_cnt);
    Initializer::initialize(init_fn, this->w, n_cnt, nxt_n_cnt);

    this->b = new Tensor(Device::Cuda, nxt_n_cnt);
    Initializer::initialize(init_fn, this->b, nxt_n_cnt, 0);

    this->dw = new Tensor(Device::Cuda, n_cnt, nxt_n_cnt);
    this->dw->reset();

    this->db = new Tensor(Device::Cuda, nxt_n_cnt);
    this->db->reset();
}

LinearLayer::~LinearLayer()
{
}

LayerType LinearLayer::get_type()
{
    return LayerType::Linear;
}

void LinearLayer::evaluate(Tensor *nxt_n, bool train_flg)
{
    int n_cnt = this->n->get_cnt();
    int nxt_n_cnt = nxt_n->get_cnt();

    // Dot product:
    {
        int threads_per_block = CUDA_THREADS_PER_BLOCK;
        int num_blocks = ((n_cnt * nxt_n_cnt) / threads_per_block) + 1;
        k_dot<<<num_blocks, threads_per_block>>>(this->n->get_arr(), w->get_arr(),
                                                 nxt_n->get_arr(), n_cnt, nxt_n_cnt);
    }

    // Add biases:
    {
        int threads_per_block(CUDA_THREADS_PER_BLOCK);
        int num_blocks((nxt_n_cnt / threads_per_block) + 1);
        k_add_bias<<<num_blocks, threads_per_block>>>(this->b->get_arr(), nxt_n->get_arr(),
                                                      nxt_n_cnt);
    }
}

Tensor *LinearLayer::derive(Tensor *dc)
{
    int dc_cnt = dc->get_cnt();
    int n_cnt = this->n->get_cnt();

    // Weights:
    {
        int threads_per_block = CUDA_THREADS_PER_BLOCK;
        int num_blocks = ((dc_cnt * n_cnt) / threads_per_block) + 1;
        k_derive_z_and_increment_weight_derivative<<<num_blocks, threads_per_block>>>(dc->get_arr(),
                                                                                      this->n->get_arr(),
                                                                                      this->dw->get_arr(),
                                                                                      dc_cnt, n_cnt);
    }

    // Biases:
    {
        int threads_per_block = CUDA_THREADS_PER_BLOCK;
        int num_blocks = (dc_cnt / threads_per_block) + 1;
        k_derive_z_and_increment_bias_derivative<<<num_blocks, threads_per_block>>>(dc->get_arr(), this->db->get_arr(), dc_cnt);
    }

    Tensor *nxt_dc = new Tensor(Device::Cuda, n_cnt);
    nxt_dc->reset();

    {
        int threads_per_block = CUDA_THREADS_PER_BLOCK;
        int num_blocks = ((n_cnt * dc_cnt) / threads_per_block) + 1;
        k_derive_z_and_aggregate_derivatives<<<num_blocks, threads_per_block>>>(dc->get_arr(), this->w->get_arr(),
                                                                                nxt_dc->get_arr(),
                                                                                dc_cnt, n_cnt);
    }

    delete dc;
    dc = nxt_dc;

    return dc;
}

void LinearLayer::step(int batch_size, float learning_rate)
{
    // Weights:
    {
        int threads_per_block = CUDA_THREADS_PER_BLOCK;
        int num_blocks = (this->w->get_cnt() / threads_per_block) + 1;
        k_adjust_weight<<<num_blocks, threads_per_block>>>(this->w->get_arr(), this->dw->get_arr(), batch_size, learning_rate,
                                                           (this->w->get_cnt()));
    }

    // Biases:
    {
        int threads_per_block = CUDA_THREADS_PER_BLOCK;
        int num_blocks = (this->b->get_cnt() / threads_per_block) + 1;
        k_adjust_bias<<<num_blocks, threads_per_block>>>(this->b->get_arr(), this->db->get_arr(), batch_size, learning_rate, this->b->get_cnt());
    }
}

void LinearLayer::load(FILE *file_ptr)
{
    int w_cnt = this->w->get_cnt();
    int b_cnt = this->b->get_cnt();

    float *w_buf = (float *)malloc(sizeof(float) * w_cnt);
    fread(w_buf, sizeof(float), w_cnt, file_ptr);
    this->w->set_arr(w_buf);
    this->w->to(Device::Cuda);
    free(w_buf);

    float *b_buf = (float *)malloc(sizeof(float) * b_cnt);
    fread(b_buf, sizeof(float), b_cnt, file_ptr);
    this->b->set_arr(b_buf);
    this->b->to(Device::Cuda);
    free(b_buf);
}

void LinearLayer::save(FILE *file_ptr)
{
    fwrite(this->w->get_arr(Device::Cpu), sizeof(float), this->w->get_cnt(), file_ptr);
    fwrite(this->b->get_arr(Device::Cpu), sizeof(float), this->b->get_cnt(), file_ptr);
}

// ActivationLayer functions:

ActivationLayer::ActivationLayer()
    : Layer() {}

ActivationLayer::ActivationLayer(int n_cnt, ActivationFunction activation_fn)
    : Layer()
{
    this->n = new Tensor(Device::Cuda, n_cnt);
    this->n->reset();

    this->activation_fn = activation_fn;
}

ActivationLayer::~ActivationLayer()
{
}

LayerType ActivationLayer::get_type()
{
    return LayerType::Activation;
}

void ActivationLayer::evaluate(Tensor *nxt_n, bool train_flg)
{
    {
        int threads_per_block = CUDA_THREADS_PER_BLOCK;
        int num_blocks = (this->n->get_cnt() / threads_per_block) + 1;
        k_activate<<<num_blocks, threads_per_block>>>(this->n->get_arr(), nxt_n->get_arr(), this->n->get_cnt(), this->activation_fn);
    }
}

Tensor *ActivationLayer::derive(Tensor *dc)
{
    {
        int threads_per_block = CUDA_THREADS_PER_BLOCK;
        int num_blocks = (this->n->get_cnt() / threads_per_block) + 1;
        k_derive_activation<<<num_blocks, threads_per_block>>>(this->n->get_arr(), dc->get_arr(), this->n->get_cnt(), this->activation_fn);
    }

    return dc;
}

void ActivationLayer::load(FILE *file_ptr)
{
    fread(&this->activation_fn, sizeof(ActivationFunction), 1, file_ptr);
}

void ActivationLayer::save(FILE *file_ptr)
{
    fwrite(&this->activation_fn, sizeof(ActivationFunction), 1, file_ptr);
}

// DropoutLayer functions:

DropoutLayer::DropoutLayer()
    : Layer() {}

DropoutLayer::DropoutLayer(int n_cnt, float dropout_rate)
{
    this->n = new Tensor(Device::Cuda, n_cnt);
    this->n->reset();

    this->dropout_rate = dropout_rate;

    this->dropout_mask = new Tensor(Device::Cuda, n_cnt);
}

DropoutLayer::~DropoutLayer()
{
    delete this->dropout_mask;
}

LayerType DropoutLayer::get_type()
{
    return LayerType::Dropout;
}

void DropoutLayer::evaluate(Tensor *nxt_n, bool train_flg)
{
    if (train_flg)
    {
        {
            int threads_per_block = CUDA_THREADS_PER_BLOCK;
            int num_blocks = (this->dropout_mask->get_cnt() / threads_per_block) + 1;
            k_set_dropout_mask<<<num_blocks, threads_per_block>>>(this->dropout_mask->get_arr(), this->dropout_mask->get_cnt(),
                                                                  this->dropout_rate);
        }

        if (this->dropout_rate > 0.0f)
        {
            int threads_per_block = CUDA_THREADS_PER_BLOCK;
            int num_blocks((nxt_n->get_cnt() / threads_per_block) + 1);
            k_dropout<<<num_blocks, threads_per_block>>>(this->n->get_arr(), this->dropout_mask->get_arr(), nxt_n->get_arr(),
                                                         nxt_n->get_cnt(), this->dropout_rate);
        }
    }
}

Tensor *DropoutLayer::derive(Tensor *dc)
{
    // Derive post-dropout activation (with respect to activation):
    {
        if (this->dropout_rate > 0.0f)
        {
            int threads_per_block = CUDA_THREADS_PER_BLOCK;
            int num_blocks = (this->n->get_cnt() / threads_per_block) + 1;
            k_derive_dropout<<<num_blocks, threads_per_block>>>(dc->get_arr(), this->dropout_mask->get_arr(),
                                                                this->n->get_cnt(), this->dropout_rate);
        }
    }

    return dc;
}

void DropoutLayer::load(FILE *file_ptr)
{
    fread(&this->dropout_rate, sizeof(float), 1, file_ptr);
}

void DropoutLayer::save(FILE *file_ptr)
{
    fwrite(&this->dropout_rate, sizeof(float), 1, file_ptr);
}