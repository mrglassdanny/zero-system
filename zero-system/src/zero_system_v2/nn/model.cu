#include "hip/hip_runtime.h"
#include "model.cuh"

using namespace zero_v2::core;
using namespace zero_v2::nn;

// Device functions:

__device__ float d_mse_cost(float n_val, float y_val)
{
    return ((n_val - y_val) * (n_val - y_val));
}

__device__ float d_derive_mse_cost(float n_val, float y_val)
{
    return 2.0f * (n_val - y_val);
}

__device__ float d_cross_entropy_cost(float n_val, float y_val)
{
    return (float)((y_val * log(n_val)) + ((1.0 - y_val) * log(1.0 - n_val)));
}

__device__ float d_derive_cross_entropy_cost(float n_val, float y_val)
{
    return (n_val - y_val);
}

// Kernel functions:

__global__ void k_cost(float *n_arr, float *y_arr, float *cost, int n_cnt, CostFunction cost_fn)
{
    __shared__ float temp[CUDA_THREADS_PER_BLOCK];
    memset(temp, 0, CUDA_THREADS_PER_BLOCK * sizeof(float));

    int tid = blockIdx.x * blockDim.x + threadIdx.x;

    if (tid < n_cnt)
    {
        switch (cost_fn)
        {
        case MSE:
            temp[threadIdx.x] = d_mse_cost(n_arr[tid], y_arr[tid]);
            break;
        case CrossEntropy:
            temp[threadIdx.x] = d_cross_entropy_cost(n_arr[tid], y_arr[tid]);
            break;
        default:
            break;
        }
    }

    __syncthreads();

    if (threadIdx.x == 0)
    {
        float sum = 0.0f;

#pragma unroll
        for (int i = 0; i < CUDA_THREADS_PER_BLOCK; i++)
        {
            sum += temp[i];
        }

        atomicAdd(cost, sum);
    }
}

__global__ void k_derive_cost(float *n_arr, float *y_arr, float *agg_derivatives_arr, int n_cnt, CostFunction cost_fn)
{
    int tid = blockIdx.x * blockDim.x + threadIdx.x;

    if (tid < n_cnt)
    {
        switch (cost_fn)
        {
        case MSE:
            agg_derivatives_arr[tid] *= d_derive_mse_cost(n_arr[tid], y_arr[tid]);
            break;
        case CrossEntropy:
            agg_derivatives_arr[tid] *= d_derive_cross_entropy_cost(n_arr[tid], y_arr[tid]);
            break;
        default:
            break;
        }
    }
}

// Model functions:

Model::Model(CostFunction cost_fn, float learning_rate)
{
    this->cost_fn = cost_fn;
    this->learning_rate = learning_rate;

    hipMalloc(&this->d_cost, sizeof(float));
    hipMemset(this->d_cost, 0, sizeof(float));
}

Model::~Model()
{
    for (Layer *lyr : this->layers)
    {
        delete lyr;
    }

    hipFree(this->d_cost);
}

void Model::load(const char *path)
{
    FILE *file_ptr = fopen(path, "rb");

    fread(&this->cost_fn, sizeof(CostFunction), 1, file_ptr);
    fread(&this->learning_rate, sizeof(float), 1, file_ptr);

    int lyr_cnt = 0;
    fread(&lyr_cnt, sizeof(int), 1, file_ptr);

    for (int i = 0; i < lyr_cnt; i++)
    {
        LayerType lyr_typ;
        fread(&lyr_typ, sizeof(LayerType), 1, file_ptr);

        Layer *lyr = nullptr;

        switch (lyr_typ)
        {
        case LayerType::Linear:
            lyr = new LinearLayer();
            break;
        case LayerType::Activation:
            lyr = new ActivationLayer();
            break;
        case LayerType::Dropout:
            lyr = new DropoutLayer();
            break;
        default:
            break;
        }

        lyr->load(file_ptr);
        this->add_layer(lyr);
    }

    hipMalloc(&this->d_cost, sizeof(float));
    hipMemset(this->d_cost, 0, sizeof(float));
}

void Model::save(const char *path)
{
    FILE *file_ptr = fopen(path, "wb");

    fwrite(&this->cost_fn, sizeof(CostFunction), 1, file_ptr);
    fwrite(&this->learning_rate, sizeof(float), 1, file_ptr);

    int lyr_cnt = this->layers.size();
    fwrite(&lyr_cnt, sizeof(int), 1, file_ptr);

    for (Layer *lyr : this->layers)
    {
        LayerType lyr_typ = lyr->get_type();
        fwrite(&lyr_typ, sizeof(LayerType), 1, file_ptr);

        lyr->save(file_ptr);
    }

    fclose(file_ptr);
}

void Model::add_layer(Layer *lyr)
{
    this->layers.push_back(lyr);
}

Tensor *Model::forward(Tensor *x)
{
    int lst_lyr_idx = this->layers.size() - 1;

    Layer *fst_lyr = this->layers[0];
    Layer *lst_lyr = this->layers[lst_lyr_idx];

    fst_lyr->n->copy(x);

    for (int i = 0; i < lst_lyr_idx; i++)
    {
        Layer *lyr = this->layers[i];
        Layer *nxt_lyr = this->layers[i + 1];

        lyr->evaluate(nxt_lyr->n);
    }

    Tensor *pred = new Tensor(Device::Cuda, lst_lyr->n->get_shape());
    lst_lyr->evaluate(pred);

    return pred;
}

float Model::cost(Tensor *pred, Tensor *y)
{
    float h_cost = 0.0f;

    {
        int threads_per_block(CUDA_THREADS_PER_BLOCK);
        int num_blocks((pred->get_cnt() / threads_per_block) + 1);

        k_cost<<<num_blocks, threads_per_block>>>(pred->get_arr(), y->get_arr(),
                                                  this->d_cost, pred->get_cnt(), this->cost_fn);
    }

    hipMemcpy(&h_cost, this->d_cost, sizeof(float), hipMemcpyDeviceToHost);

    hipMemset(this->d_cost, 0, sizeof(float));

    return h_cost;
}

void Model::backward(Tensor *pred, Tensor *y)
{
    Tensor *dc = new Tensor(Device::Cuda, pred->get_shape());
    dc->set_all(1.0f);

    // Derive cost (with respect to activation):
    {
        int threads_per_block = CUDA_THREADS_PER_BLOCK;
        int num_blocks = (pred->get_cnt() / threads_per_block) + 1;
        k_derive_cost<<<num_blocks, threads_per_block>>>(pred->get_arr(),
                                                         y->get_arr(), dc->get_arr(), pred->get_cnt(), this->cost_fn);
    }

    int lst_lyr_idx = this->layers.size() - 1;

    for (int i = lst_lyr_idx; i > 0; i--)
    {
        Layer *lyr = this->layers[i];
        lyr->derive(dc);
    }

    delete dc;
}

void Model::step(int batch_size)
{
    for (Layer *lyr : this->layers)
    {
        if (LearnableLayer *lrn_lyr = dynamic_cast<LearnableLayer *>(lyr))
        {
            lrn_lyr->step(batch_size, this->learning_rate);
        }
    }
}
