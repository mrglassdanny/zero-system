#include "hip/hip_runtime.h"
#include "model.cuh"

using namespace zero_v2::core;
using namespace zero_v2::nn;

// Device functions:

__device__ float d_mse_cost(float n_val, float y_val)
{
    return ((n_val - y_val) * (n_val - y_val));
}

__device__ float d_derive_mse_cost(float n_val, float y_val)
{
    return 2.0f * (n_val - y_val);
}

__device__ float d_cross_entropy_cost(float n_val, float y_val)
{
    return (float)((y_val * log(n_val)) + ((1.0 - y_val) * log(1.0 - n_val)));
}

__device__ float d_derive_cross_entropy_cost(float n_val, float y_val)
{
    return (n_val - y_val);
}

// Kernel functions:

__global__ void k_cost(float *n_arr, float *y_arr, float *cost, int n_cnt, CostFunction cost_fn)
{
    __shared__ float temp[CUDA_THREADS_PER_BLOCK];
    memset(temp, 0, CUDA_THREADS_PER_BLOCK * sizeof(float));

    int tid = blockIdx.x * blockDim.x + threadIdx.x;

    if (tid < n_cnt)
    {
        switch (cost_fn)
        {
        case MSE:
            temp[threadIdx.x] = d_mse_cost(n_arr[tid], y_arr[tid]);
            break;
        case CrossEntropy:
            temp[threadIdx.x] = d_cross_entropy_cost(n_arr[tid], y_arr[tid]);
            break;
        default:
            break;
        }
    }

    __syncthreads();

    if (threadIdx.x == 0)
    {
        float sum = 0.0f;

#pragma unroll
        for (int i = 0; i < CUDA_THREADS_PER_BLOCK; i++)
        {
            sum += temp[i];
        }

        atomicAdd(cost, sum);
    }
}

__global__ void k_derive_cost(float *n_arr, float *y_arr, float *agg_derivatives_arr, int n_cnt, CostFunction cost_fn)
{
    int tid = blockIdx.x * blockDim.x + threadIdx.x;

    if (tid < n_cnt)
    {
        switch (cost_fn)
        {
        case MSE:
            agg_derivatives_arr[tid] *= d_derive_mse_cost(n_arr[tid], y_arr[tid]);
            break;
        case CrossEntropy:
            agg_derivatives_arr[tid] *= d_derive_cross_entropy_cost(n_arr[tid], y_arr[tid]);
            break;
        default:
            break;
        }
    }
}

// Model functions:

Model::Model(CostFunction cost_fn, float learning_rate)
{
    this->cost_fn = cost_fn;
    this->learning_rate = learning_rate;

    hipMalloc(&this->d_cost, sizeof(float));
    hipMemset(this->d_cost, 0, sizeof(float));
}

Model::Model(const char *path)
{
    FILE *file_ptr = fopen(path, "rb");

    fread(&this->cost_fn, sizeof(CostFunction), 1, file_ptr);
    fread(&this->learning_rate, sizeof(float), 1, file_ptr);

    int lyr_cnt = 0;
    fread(&lyr_cnt, sizeof(int), 1, file_ptr);

    for (int i = 0; i < lyr_cnt; i++)
    {
        LayerType lyr_typ;
        fread(&lyr_typ, sizeof(LayerType), 1, file_ptr);

        Layer *lyr = nullptr;

        switch (lyr_typ)
        {
        case LayerType::Linear:
            lyr = new LinearLayer();
            break;
        case LayerType::Convolutional:
            lyr = new ConvolutionalLayer();
            break;
        case LayerType::Activation:
            lyr = new ActivationLayer();
            break;
        case LayerType::Dropout:
            lyr = new DropoutLayer();
            break;
        default:
            break;
        }

        lyr->load(file_ptr);
        this->add_layer(lyr);
    }

    hipMalloc(&this->d_cost, sizeof(float));
    hipMemset(this->d_cost, 0, sizeof(float));

    fclose(file_ptr);
}

Model::~Model()
{
    for (Layer *lyr : this->layers)
    {
        delete lyr;
    }

    hipFree(this->d_cost);
}

void Model::save(const char *path)
{
    FILE *file_ptr = fopen(path, "wb");

    fwrite(&this->cost_fn, sizeof(CostFunction), 1, file_ptr);
    fwrite(&this->learning_rate, sizeof(float), 1, file_ptr);

    int lyr_cnt = this->layers.size();
    fwrite(&lyr_cnt, sizeof(int), 1, file_ptr);

    for (Layer *lyr : this->layers)
    {
        LayerType lyr_typ = lyr->get_type();
        fwrite(&lyr_typ, sizeof(LayerType), 1, file_ptr);

        lyr->save(file_ptr);
    }

    fclose(file_ptr);
}

void Model::add_layer(Layer *lyr)
{
    this->layers.push_back(lyr);
}

std::vector<int> Model::get_input_shape()
{
    return this->layers[0]->get_input_shape();
}

std::vector<int> Model::get_output_shape()
{
    return this->layers[this->layers.size() - 1]->get_output_shape();
}

Tensor *Model::forward(Tensor *x, bool train_flg)
{
    int lst_lyr_idx = this->layers.size() - 1;

    Layer *frst_lyr = this->layers[0];
    Layer *lst_lyr = this->layers[lst_lyr_idx];

    frst_lyr->n->copy(x);

    for (int i = 0; i < lst_lyr_idx; i++)
    {
        Layer *lyr = this->layers[i];
        Layer *nxt_lyr = this->layers[i + 1];

        lyr->evaluate(nxt_lyr->n, train_flg);
    }

    Tensor *pred = new Tensor(Device::Cuda, lst_lyr->n->get_shape());
    lst_lyr->evaluate(pred, train_flg);

    return pred;
}

float Model::cost(Tensor *pred, Tensor *y)
{
    float h_cost = 0.0f;

    {
        int threads_per_block = CUDA_THREADS_PER_BLOCK;
        int num_blocks = (pred->get_cnt() / threads_per_block) + 1;

        k_cost<<<num_blocks, threads_per_block>>>(pred->get_arr(), y->get_arr(),
                                                  this->d_cost, pred->get_cnt(), this->cost_fn);
    }

    hipMemcpy(&h_cost, this->d_cost, sizeof(float), hipMemcpyDeviceToHost);

    hipMemset(this->d_cost, 0, sizeof(float));

    return h_cost;
}

void Model::backward(Tensor *pred, Tensor *y)
{
    Tensor *dc = new Tensor(Device::Cuda, pred->get_shape());
    dc->set_all(1.0f);

    {
        int threads_per_block = CUDA_THREADS_PER_BLOCK;
        int num_blocks = (pred->get_cnt() / threads_per_block) + 1;
        k_derive_cost<<<num_blocks, threads_per_block>>>(pred->get_arr(),
                                                         y->get_arr(), dc->get_arr(), pred->get_cnt(), this->cost_fn);
    }

    int lst_lyr_idx = this->layers.size() - 1;

    for (int i = lst_lyr_idx; i >= 0; i--)
    {
        Layer *lyr = this->layers[i];
        dc = lyr->derive(dc);
    }

    delete dc;
}

void Model::step(int batch_size)
{
    for (Layer *lyr : this->layers)
    {
        if (LearnableLayer *lrn_lyr = dynamic_cast<LearnableLayer *>(lyr))
        {
            lrn_lyr->step(batch_size, this->learning_rate);
        }
    }
}

void Model::gradient_check(Tensor *x, Tensor *y, bool print_flg)
{
    float agg_ana_grad = 0.0f;
    float agg_num_grad = 0.0f;
    float agg_grad_diff = 0.0f;

    float epsilon = 0.001f;

    // Analytical gradients:
    {
        Tensor *pred = this->forward(x, true);
        this->backward(pred, y);
        delete pred;
    }

    // Numerical gradients:
    {
        int lyr_idx = 0;

        for (Layer *lyr : this->layers)
        {
            lyr_idx++;

            if (LearnableLayer *lrn_lyr = dynamic_cast<LearnableLayer *>(lyr))
            {
                for (int i = 0; i < lrn_lyr->w->get_cnt(); i++)
                {
                    float left_cost = 0.0;
                    float right_cost = 0.0;

                    float orig_w_val = lrn_lyr->w->get_val(i);

                    float left_w_val = orig_w_val - epsilon;
                    float right_w_val = orig_w_val + epsilon;

                    float ana_grad = lrn_lyr->dw->get_val(i);

                    lrn_lyr->w->set_val(i, left_w_val);
                    {
                        Tensor *pred = this->forward(x, true);
                        left_cost = this->cost(pred, y);
                        delete pred;
                    }

                    lrn_lyr->w->set_val(i, right_w_val);
                    {
                        Tensor *pred = this->forward(x, true);
                        right_cost = this->cost(pred, y);
                        delete pred;
                    }

                    float num_grad = (right_cost - left_cost) / (2.0f * epsilon);

                    if (print_flg)
                    {
                        printf("W: %d  %d\t%f : %f  (%f)\n", lyr_idx, i, ana_grad, num_grad, fabs(ana_grad - num_grad));
                    }

                    agg_ana_grad += (ana_grad * ana_grad);
                    agg_num_grad += (num_grad * num_grad);
                    agg_grad_diff += ((ana_grad - num_grad) * (ana_grad - num_grad));

                    lrn_lyr->w->set_val(i, orig_w_val);
                }

                for (int i = 0; i < lrn_lyr->b->get_cnt(); i++)
                {
                    float left_cost = 0.0;
                    float right_cost = 0.0;

                    float orig_b_val = lrn_lyr->b->get_val(i);

                    float left_b_val = orig_b_val - epsilon;
                    float right_b_val = orig_b_val + epsilon;

                    float ana_grad = lrn_lyr->db->get_val(i);

                    lrn_lyr->b->set_val(i, left_b_val);
                    {
                        Tensor *pred = this->forward(x, true);
                        left_cost = this->cost(pred, y);
                        delete pred;
                    }

                    lrn_lyr->b->set_val(i, right_b_val);
                    {
                        Tensor *pred = this->forward(x, true);
                        right_cost = this->cost(pred, y);
                        delete pred;
                    }

                    float num_grad = (right_cost - left_cost) / (2.0f * epsilon);

                    if (print_flg)
                    {
                        printf("B: %d  %d\t%f : %f  (%f)\n", lyr_idx, i, ana_grad, num_grad, fabs(ana_grad - num_grad));
                    }

                    agg_ana_grad += (ana_grad * ana_grad);
                    agg_num_grad += (num_grad * num_grad);
                    agg_grad_diff += ((ana_grad - num_grad) * (ana_grad - num_grad));

                    lrn_lyr->b->set_val(i, orig_b_val);
                }
            }
        }
    }

    if ((agg_grad_diff) == 0.0f && (agg_ana_grad + agg_num_grad) == 0.0f)
    {
        printf("GRADIENT CHECK RESULT: %f\n", 0.0f);
    }
    else
    {
        printf("GRADIENT CHECK RESULT: %f\n", (agg_grad_diff) / (agg_ana_grad + agg_num_grad));
    }
}