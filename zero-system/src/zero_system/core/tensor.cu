#include "Tensor.cuh"

using namespace zero::core;

Tensor *Tensor::one_hot_encode(int row_cnt, int col_cnt, TensorType typ, float *cpu_arr)
{
    Tensor *tensor = new Tensor(row_cnt, col_cnt, typ);
    tensor->set_all(0.0f);

    for (int i = 0; i < row_cnt; i++)
    {
        int col_idx = (int)cpu_arr[i];
        if (col_idx < col_cnt)
        {
            tensor->set_rowcol(i, col_idx, 1.0f);
        }
        // If column index is greater than or equal to column count, skip it!
        // ^ this shouldn't happen...
    }

    return tensor;
}

// Geared toward small csv files (under 0.5 GB).
Tensor *Tensor::from_csv(const char *csv_file_name)
{
    FILE *file_ptr = fopen(csv_file_name, "rb");

    fseek(file_ptr, 0L, SEEK_END);
    long file_size = ftell(file_ptr);
    rewind(file_ptr);

    char *buf = (char *)malloc(file_size + 1);
    memset(buf, 0, file_size + 1);
    fread(buf, 1, file_size, file_ptr);

    fclose(file_ptr);

    int buf_idx = 0;

    int row_cnt = 0;
    int col_cnt = 0;

    while (buf[buf_idx] != '\n')
    {
        if (buf[buf_idx] == ',')
        {
            col_cnt++;
        }

        buf_idx++;
    }

    col_cnt++;
    buf_idx++;

    int lst_row_idx = 0;
    for (int i = buf_idx; i < file_size; i++)
    {
        if (buf[i] == '\n')
        {
            row_cnt++;
            lst_row_idx = i;
        }
    }

    // If file does not end in newline, add to the row count.
    if (lst_row_idx < file_size - 1)
    {
        row_cnt++;
    }

    Tensor *tensor = new Tensor(row_cnt, col_cnt, Cpu);

    char temp_buf[64];
    memset(temp_buf, 0, 64);
    int temp_buf_idx = 0;
    int row_idx = 0;
    int col_idx = 0;

    for (; buf_idx < file_size; buf_idx++)
    {
        while (buf[buf_idx] != ',' && buf[buf_idx] != '\n' && buf_idx < file_size)
        {
            if (buf[buf_idx] != '"')
            {
                temp_buf[temp_buf_idx++] = buf[buf_idx];
            }

            buf_idx++;
        }

        if (buf[buf_idx] == ',')
        {
            tensor->set_rowcol(row_idx, col_idx, (float)atof(temp_buf));
            memset(temp_buf, 0, 64);
            col_idx++;
            temp_buf_idx = 0;
        }
        else if (buf[buf_idx] == '\n')
        {
            tensor->set_rowcol(row_idx, col_idx, (float)atof(temp_buf));
            memset(temp_buf, 0, 64);
            row_idx++;
            col_idx = 0;
            temp_buf_idx = 0;
        }
    }

    // Make sure to grab the last bit before we finish up!
    if (temp_buf_idx > 0)
    {
        tensor->set_rowcol(row_idx, col_idx, (float)atof(temp_buf));
        memset(temp_buf, 0, 64);
        row_idx++;
        col_idx = 0;
        temp_buf_idx = 0;
    }

    free(buf);

    return tensor;
}

Tensor::Tensor(int row_cnt, int col_cnt, TensorType typ)
{
    if (typ == Gpu)
    {
        hipMalloc(&this->arr, sizeof(float) * (row_cnt * col_cnt));
    }
    else
    {
        this->arr = (float *)malloc(sizeof(float) * (row_cnt * col_cnt));
    }

    this->row_cnt = row_cnt;
    this->col_cnt = col_cnt;
    this->typ = typ;
}

Tensor::Tensor(const Tensor &src)
{
    if (src.typ == Gpu)
    {
        hipMalloc(&this->arr, sizeof(float) * (src.row_cnt * src.col_cnt));
        hipMemcpy(this->arr, src.arr, sizeof(float) * (src.row_cnt * src.col_cnt), hipMemcpyDeviceToDevice);
    }
    else
    {
        this->arr = (float *)malloc(sizeof(float) * (src.row_cnt * src.col_cnt));
        memcpy(this->arr, src.arr, sizeof(float) * (src.row_cnt * src.col_cnt));
    }

    this->row_cnt = src.row_cnt;
    this->col_cnt = src.col_cnt;
    this->typ = src.typ;
}

Tensor::Tensor(const Tensor &src, TensorType typ)
{

    if (src.typ == Gpu && typ == Gpu)
    {
        hipMalloc(&this->arr, sizeof(float) * (src.row_cnt * src.col_cnt));
        hipMemcpy(this->arr, src.arr, sizeof(float) * (src.row_cnt * src.col_cnt), hipMemcpyDeviceToDevice);
    }
    else if (src.typ == Cpu && typ == Cpu)
    {
        this->arr = (float *)malloc(sizeof(float) * (src.row_cnt * src.col_cnt));
        memcpy(this->arr, src.arr, sizeof(float) * (src.row_cnt * src.col_cnt));
    }
    else if (src.typ == Cpu && typ == Gpu)
    {
        hipMalloc(&this->arr, sizeof(float) * (src.row_cnt * src.col_cnt));
        hipMemcpy(this->arr, src.arr, sizeof(float) * (src.row_cnt * src.col_cnt), hipMemcpyHostToDevice);
    }
    else if (src.typ == Gpu && typ == Cpu)
    {
        this->arr = (float *)malloc(sizeof(float) * (src.row_cnt * src.col_cnt));
        hipMemcpy(this->arr, src.arr, sizeof(float) * (src.row_cnt * src.col_cnt), hipMemcpyDeviceToHost);
    }

    this->row_cnt = src.row_cnt;
    this->col_cnt = src.col_cnt;
    this->typ = typ;
}

Tensor::Tensor(int row_cnt, int col_cnt, TensorType typ, float *cpu_arr)
{
    if (typ == Gpu)
    {
        hipMalloc(&this->arr, sizeof(float) * (row_cnt * col_cnt));
        hipMemcpy(this->arr, cpu_arr, sizeof(float) * (row_cnt * col_cnt), hipMemcpyHostToDevice);
    }
    else
    {
        this->arr = (float *)malloc(sizeof(float) * (row_cnt * col_cnt));
        memcpy(this->arr, cpu_arr, sizeof(float) * (row_cnt * col_cnt));
    }

    this->row_cnt = row_cnt;
    this->col_cnt = col_cnt;
    this->typ = typ;
}

Tensor::Tensor(int row_cnt, int col_cnt, TensorType typ, int *cpu_arr)
{

    if (typ == Gpu)
    {
        hipMalloc(&this->arr, sizeof(float) * (row_cnt * col_cnt));

        for (int i = 0; i < row_cnt * col_cnt; i++)
        {
            float f = (float)cpu_arr[i];
            hipMemcpy(&this->arr[i], &f, sizeof(float), hipMemcpyHostToDevice);
        }
    }
    else
    {
        this->arr = (float *)malloc(sizeof(float) * (row_cnt * col_cnt));

        for (int i = 0; i < row_cnt * col_cnt; i++)
        {
            this->arr[i] = (float)cpu_arr[i];
        }
    }

    this->row_cnt = row_cnt;
    this->col_cnt = col_cnt;
    this->typ = typ;
}

Tensor::~Tensor()
{
    if (this->typ == Gpu)
    {
        hipFree(this->arr);
    }
    else
    {
        free(this->arr);
    }
}

void Tensor::print()
{
    TensorType orig_typ = this->typ;

    this->translate(Cpu);

    {
        printf("[");
        for (int i = 0; i < this->row_cnt; i++)
        {

            if (i == 0)
            {
                printf(" [ ");
            }
            else
            {
                printf("  [ ");
            }

            for (int j = 0; j < this->col_cnt; j++)
            {
                if (j == this->col_cnt - 1)
                {
                    printf("%f", this->arr[i * this->col_cnt + j]);
                }
                else
                {
                    printf("%f, ", this->arr[i * this->col_cnt + j]);
                }
            }

            if (i == this->row_cnt - 1)
            {
                printf(" ] ");
            }
            else
            {
                printf(" ],\n");
            }
        }
        printf("]\n");
    }

    this->translate(orig_typ);
}

void Tensor::dump_to_csv(const char *csv_file_name)
{
    FILE *file_ptr = fopen(csv_file_name, "w");

    for (int j = 0; j < this->col_cnt; j++)
    {

        if (j < this->col_cnt - 1)
        {
            fprintf(file_ptr, "col_%d,", j);
        }
        else
        {
            fprintf(file_ptr, "col_%d", j);
        }
    }
    fprintf(file_ptr, "\n");

    for (int i = 0; i < this->row_cnt; i++)
    {
        for (int j = 0; j < this->col_cnt; j++)
        {
            if (j < this->col_cnt - 1)
            {
                fprintf(file_ptr, "%f,", this->get_rowcol(i, j));
            }
            else
            {
                fprintf(file_ptr, "%f", this->get_rowcol(i, j));
            }
        }
        fprintf(file_ptr, "\n");
    }
    fclose(file_ptr);
}

void Tensor::translate(TensorType typ)
{
    if (typ == Gpu)
    {
        if (this->typ != Gpu)
        {
            float *d_arr;
            hipMalloc(&d_arr, sizeof(float) * (this->row_cnt * this->col_cnt));
            hipMemcpy(d_arr, this->arr, sizeof(float) * (this->row_cnt * this->col_cnt), hipMemcpyHostToDevice);
            free(this->arr);
            this->arr = d_arr;

            this->typ = Gpu;
        }
    }
    else
    {
        if (this->typ == Gpu)
        {
            float *h_arr = (float *)malloc(sizeof(float) * (this->row_cnt * this->col_cnt));
            hipMemcpy(h_arr, this->arr, sizeof(float) * (this->row_cnt * this->col_cnt), hipMemcpyDeviceToHost);
            hipFree(this->arr);
            this->arr = h_arr;

            this->typ = Cpu;
        }
    }
}

int Tensor::get_row_cnt()
{
    return this->row_cnt;
}

int Tensor::get_col_cnt()
{
    return this->col_cnt;
}

float Tensor::get_idx(int idx)
{
    if (this->typ == Gpu)
    {
        float val;
        hipMemcpy(&val, &this->arr[idx], sizeof(float), hipMemcpyDeviceToHost);
        return val;
    }
    else
    {
        return this->arr[idx];
    }
}

float Tensor::get_rowcol(int row_idx, int col_idx)
{
    int idx = row_idx * this->col_cnt + col_idx;
    return this->get_idx(idx);
}

float *Tensor::get_arr(TensorType typ)
{
    this->translate(typ);
    return this->arr;
}

float *Tensor::get_slice(int idx, TensorType typ)
{
    this->translate(typ);
    return &this->arr[idx];
}

TensorTuple Tensor::get_min()
{
    TensorTuple tup;

    tup.idx = 0;
    tup.val = FLT_MAX;

    for (int i = 0; i < this->row_cnt * this->col_cnt; i++)
    {
        float cur_val = this->get_idx(i);
        if (cur_val < tup.val)
        {
            tup.idx = i;
            tup.val = cur_val;
        }
    }

    return tup;
}

TensorTuple Tensor::get_max()
{
    TensorTuple tup;

    tup.idx = 0;
    tup.val = -FLT_MAX;

    for (int i = 0; i < this->row_cnt * this->col_cnt; i++)
    {
        float cur_val = this->get_idx(i);
        if (cur_val > tup.val)
        {
            tup.idx = i;
            tup.val = cur_val;
        }
    }

    return tup;
}

void Tensor::set_idx(int idx, float val)
{
    if (this->typ == Gpu)
    {
        hipMemcpy(&this->arr[idx], &val, sizeof(float), hipMemcpyHostToDevice);
    }
    else
    {
        this->arr[idx] = val;
    }
}

void Tensor::set_rowcol(int row_idx, int col_idx, float val)
{
    int idx = row_idx * this->col_cnt + col_idx;
    return this->set_idx(idx, val);
}

void Tensor::set_all(float val)
{
    int tot_cnt = this->row_cnt * this->col_cnt;

    TensorType orig_typ = this->typ;

    this->translate(Cpu);

    for (int i = 0; i < tot_cnt; i++)
    {
        this->arr[i] = val;
    }

    this->translate(orig_typ);
}

void Tensor::set_all_rand(float upper)
{
    int tot_cnt = this->row_cnt * this->col_cnt;

    TensorType orig_typ = this->typ;

    this->translate(Cpu);

    for (int i = 0; i < tot_cnt; i++)
    {
        float val = (float)rand() / ((float)RAND_MAX);
        val *= (2 * upper);
        val -= upper;
        this->arr[i] = val;
    }

    this->translate(orig_typ);
}

void Tensor::set_arr(float *cpu_arr)
{
    int tot_cnt = this->row_cnt * this->col_cnt;

    TensorType orig_typ = this->typ;

    this->translate(Cpu);

    memcpy(this->arr, cpu_arr, sizeof(float) * tot_cnt);

    this->translate(orig_typ);
}
