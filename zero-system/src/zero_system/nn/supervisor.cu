#include "hip/hip_runtime.h"
#include "supervisor.cuh"

using namespace zero::nn;
using namespace zero::core;

Record::Record(Tensor *x, Tensor *y)
{
    this->x = x;
    this->y = y;
}

Record::~Record()
{
    delete this->x;
    delete this->y;
}

Batch::Batch()
{
}

Batch::~Batch()
{
}

void Batch::add(Record *record)
{
    this->records.push_back(record);
}

int Batch::get_size()
{
    return this->records.size();
}

Tensor *Batch::get_x(int idx)
{
    return this->records[idx]->x;
}

Tensor *Batch::get_y(int idx)
{
    return this->records[idx]->y;
}

Supervisor::Supervisor()
{
}

Supervisor::Supervisor(int row_cnt, int col_cnt, int one_hot_cnt, float *x_arr, float *y_arr, TensorType typ)
{
    this->add_all(row_cnt, col_cnt, one_hot_cnt, x_arr, y_arr, typ);
}

Supervisor::~Supervisor()
{
    this->clear();
}

void Supervisor::add(int col_cnt, int one_hot_cnt, float *x_arr, float y_val, TensorType typ)
{
    Tensor *x = new Tensor(1, col_cnt, typ, x_arr);

    Tensor *y;
    // Single value or one hot encoded?
    if (one_hot_cnt > 1)
    {
        // One hot encode!
        y = Tensor::one_hot_encode(1, one_hot_cnt, typ, &y_val);
    }
    else
    {
        // Single value.
        y = new Tensor(1, 1, typ, &y_val);
    }

    this->records.push_back(Record(x, y));
}

void Supervisor::add_all(int row_cnt, int col_cnt, int one_hot_cnt, float *x_arr, float *y_arr, TensorType typ)
{
    this->records.reserve(row_cnt);

    for (int i = 0; i < row_cnt; i++)
    {
        this->add(col_cnt, one_hot_cnt, &x_arr[i * col_cnt], y_arr[i], typ);
    }
}

void Supervisor::clear()
{
    this->records.clear();
}

int Supervisor::get_cnt()
{
    return this->records.size();
}

void Supervisor::shuffle()
{
    auto rng = std::default_random_engine{};
    std::shuffle(std::begin(this->records), std::end(this->records), rng);
}

// Creates batch with all data
Batch *Supervisor::create_batch()
{
    int cnt = this->get_cnt();

    if (cnt == 0)
    {
        return nullptr;
    }

    Batch *batch = new Batch();

    for (int i = 0; i < cnt; i++)
    {
        batch->add(&this->records[i]);
    }

    return batch;
}

Batch *Supervisor::create_batch(int lower, int upper)
{
    if (this->get_cnt() == 0)
    {
        return nullptr;
    }

    Batch *batch = new Batch();

    for (int i = lower; i < upper; i++)
    {
        batch->add(&this->records[i]);
    }

    return batch;
}

// Creates batch of specified size and bounds with random records.
Batch *Supervisor::create_batch(int batch_size, int lower, int upper)
{
    if (this->get_cnt() == 0)
    {
        return nullptr;
    }

    Batch *batch = new Batch();

    for (int i = 0; i < batch_size; i++)
    {
        int idx = (rand() % (upper - lower)) + lower;
        batch->add(&this->records[i]);
    }

    return batch;
}

Batch *Supervisor::create_train_batch()
{
    return this->create_batch(0, (int)floor(this->records.size() * SUPERVISOR_TRAIN_SPLIT));
}

Batch *Supervisor::create_train_batch(int batch_size)
{
    return this->create_batch(batch_size, 0, (int)floor(this->records.size() * SUPERVISOR_TRAIN_SPLIT));
}

Batch *Supervisor::create_validation_batch()
{
    return this->create_batch((int)floor(this->records.size() * SUPERVISOR_TRAIN_SPLIT), (int)floor(this->records.size() *
                                                                                                    (SUPERVISOR_TRAIN_SPLIT + SUPERVISOR_VALIDATION_SPLIT)));
}

Batch *Supervisor::create_test_batch()
{
    return this->create_batch((int)floor(this->records.size() * (SUPERVISOR_TRAIN_SPLIT + SUPERVISOR_VALIDATION_SPLIT)), this->records.size());
}