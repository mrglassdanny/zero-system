#include "hip/hip_runtime.h"
#include "supervisor.cuh"

using namespace zero::nn;
using namespace zero::core;

Batch::Batch()
{
}

Batch::~Batch()
{
}

void Batch::add(Tensor *x, Tensor *y)
{
    this->xs.push_back(x);
    this->ys.push_back(y);
}

int Batch::get_size()
{
    return this->xs.size();
}

Tensor *Batch::get_x(int idx)
{
    return this->xs[idx];
}

Tensor *Batch::get_y(int idx)
{
    return this->ys[idx];
}

Supervisor::Supervisor()
{
}

Supervisor::Supervisor(int row_cnt, int col_cnt, int one_hot_cnt, float *x_arr, float *y_arr, TensorType typ)
{
    this->add_all(row_cnt, col_cnt, one_hot_cnt, x_arr, y_arr, typ);
}

Supervisor::~Supervisor()
{
    this->clear();
}

void Supervisor::add(int col_cnt, int one_hot_cnt, float *x_arr, float y_val, TensorType typ)
{
    Tensor *x = new Tensor(1, col_cnt, typ, x_arr);

    Tensor *y;
    // Single value or one hot encoded?
    if (one_hot_cnt > 1)
    {
        // One hot encode!
        y = Tensor::one_hot_encode(1, one_hot_cnt, typ, &y_val);
    }
    else
    {
        // Single value.
        y = new Tensor(1, 1, typ, &y_val);
    }

    this->xs.push_back(x);
    this->ys.push_back(y);
}

void Supervisor::add_all(int row_cnt, int col_cnt, int one_hot_cnt, float *x_arr, float *y_arr, TensorType typ)
{
    this->xs.reserve(row_cnt);
    this->ys.reserve(row_cnt);

    for (int i = 0; i < row_cnt; i++)
    {
        this->add(col_cnt, one_hot_cnt, &x_arr[i * col_cnt], y_arr[i], typ);
    }
}

void Supervisor::clear()
{
    for (int i = 0; i < this->xs.size(); i++)
    {
        delete this->xs[i];
        delete this->ys[i];
    }

    this->xs.clear();
    this->ys.clear();
}

int Supervisor::get_cnt()
{
    return this->xs.size();
}

// Creates batch with all data
Batch *Supervisor::create_batch()
{
    int cnt = this->get_cnt();

    if (cnt == 0)
    {
        return nullptr;
    }

    Batch *batch = new Batch();

    for (int i = 0; i < cnt; i++)
    {
        batch->add(this->xs[i], this->ys[i]);
    }

    return batch;
}

Batch *Supervisor::create_batch(int lower, int upper)
{
    if (this->get_cnt() == 0)
    {
        return nullptr;
    }

    Batch *batch = new Batch();

    for (int i = lower; i < upper; i++)
    {
        batch->add(this->xs[i], this->ys[i]);
    }

    return batch;
}

Batch *Supervisor::create_batch(int batch_size, int lower, int upper)
{
    if (this->get_cnt() == 0)
    {
        return nullptr;
    }

    Batch *batch = new Batch();

    for (int i = 0; i < batch_size; i++)
    {
        int idx = (rand() % (upper - lower)) + lower;
        batch->add(this->xs[idx], this->ys[idx]);
    }

    return batch;
}

Batch *Supervisor::create_train_batch()
{
    return this->create_batch(0, (int)floor(this->xs.size() * SUPERVISOR_TRAIN_SPLIT));
}

Batch *Supervisor::create_train_batch(int batch_size)
{
    return this->create_batch(batch_size, 0, (int)floor(this->xs.size() * SUPERVISOR_TRAIN_SPLIT));
}

Batch *Supervisor::create_validation_batch()
{
    return this->create_batch((int)floor(this->xs.size() * SUPERVISOR_TRAIN_SPLIT), (int)floor(this->xs.size() *
                                                                                               (SUPERVISOR_TRAIN_SPLIT + SUPERVISOR_VALIDATION_SPLIT)));
}

Batch *Supervisor::create_test_batch()
{
    return this->create_batch((int)floor(this->xs.size() * (SUPERVISOR_TRAIN_SPLIT + SUPERVISOR_VALIDATION_SPLIT)), this->xs.size());
}