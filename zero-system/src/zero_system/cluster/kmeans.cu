#include "hip/hip_runtime.h"
#include "kmeans.cuh"

#define THREADS_PER_BLOCK 32
#define MAX_CLUSTER_CNT 128

using namespace zero::core;
using namespace zero::cluster;

// Device functions:

float __device__ d_get_cost(float x_val, float cluster_val)
{
    return ((x_val - cluster_val) * (x_val - cluster_val));
}

int __device__ d_get_min(float *arr, int cnt)
{
    int min_idx = 0;
    float min_val = FLT_MAX;

    for (int i = 0; i < cnt; i++)
    {
        float cur_val = arr[i];
        if (cur_val < min_val)
        {
            min_idx = i;
            min_val = cur_val;
        }
    }

    return min_idx;
}

// Kernel functions:

void __global__ k_reset(float *cluster_arr, int cnt)
{
    int tid = blockIdx.x * blockDim.x + threadIdx.x;

    if (tid < cnt)
    {
        cluster_arr[tid] = 0.0f;
    }
}

void __global__ k_assign(float *x_arr, float *assignment_arr, float *cluster_arr, float *cost, int feature_cnt, int cluster_cnt, int row_cnt)
{
    float temp[MAX_CLUSTER_CNT];
    memset(temp, 0, sizeof(float) * MAX_CLUSTER_CNT);

    int tid = blockIdx.x * blockDim.x + threadIdx.x;

    if (tid < row_cnt)
    {
        for (int feature_idx = 0; feature_idx < feature_cnt; feature_idx++)
        {
            for (int cluster_idx = 0; cluster_idx < cluster_cnt; cluster_idx++)
            {
                temp[cluster_idx] += d_get_cost(x_arr[tid * feature_cnt + feature_idx], cluster_arr[cluster_idx * feature_cnt + feature_idx]);
            }
        }

        for (int cluster_idx = 0; cluster_idx < cluster_cnt; cluster_idx++)
        {
            temp[cluster_idx] = sqrt(temp[cluster_idx]);
        }

        int min_cluster_idx = d_get_min(temp, cluster_cnt);

        assignment_arr[tid] = min_cluster_idx;

        if (cost != nullptr)
        {
            atomicAdd(cost, temp[min_cluster_idx]);
        }
    }
}

void __global__ k_update_part_1(float *x_arr, float *assignment_arr, float *cluster_arr, float *assignment_cnt_arr, int feature_cnt, int cluster_cnt, int row_cnt)
{
    int tid = blockIdx.x * blockDim.x + threadIdx.x;

    if (tid < row_cnt)
    {
        int cluster_idx = assignment_arr[tid];

        atomicAdd(&assignment_cnt_arr[cluster_idx], 1.0f);

        for (int feature_idx = 0; feature_idx < feature_cnt; feature_idx++)
        {
            atomicAdd(&cluster_arr[cluster_idx * feature_cnt + feature_idx], x_arr[tid * feature_cnt + feature_idx]);
        }
    }
}

void __global__ k_update_part_2(float *cluster_arr, float *assignment_cnt_arr, int cluster_cnt, int feature_cnt)
{
    int tid = blockIdx.x * blockDim.x + threadIdx.x;

    if (tid < cluster_cnt * feature_cnt)
    {
        int cluster_idx = tid / feature_cnt;

        cluster_arr[tid] /= (assignment_cnt_arr[cluster_idx]);
    }
}

// KMeans member functions:

KMeans::KMeans(int cluster_cnt, int feature_cnt)
{
    this->cluster_cnt = cluster_cnt;
    this->feature_cnt = feature_cnt;

    this->clusters = new Tensor(cluster_cnt, feature_cnt, Cpu);
}

KMeans::KMeans(const KMeans &src)
{
    this->cluster_cnt = src.cluster_cnt;
    this->feature_cnt = src.feature_cnt;
    this->clusters = new Tensor(*src.clusters);
}

KMeans::KMeans(const char *path)
{
    FILE *file_ptr = fopen(path, "rb");

    fread(&this->cluster_cnt, sizeof(int), 1, file_ptr);
    fread(&this->feature_cnt, sizeof(int), 1, file_ptr);

    int tot_cnt = (this->cluster_cnt * this->feature_cnt);
    float *cluster_buf = (float *)malloc(sizeof(float) * tot_cnt);
    fread(cluster_buf, sizeof(float), tot_cnt, file_ptr);
    this->clusters = new Tensor(this->cluster_cnt, this->feature_cnt, Gpu, cluster_buf);
    free(cluster_buf);

    fclose(file_ptr);
}

KMeans::~KMeans()
{
    delete this->clusters;
}

void KMeans::print()
{
    this->clusters->print();
}

void KMeans::dump(const char *path)
{
    FILE *file_ptr = fopen(path, "wb");

    fwrite(&this->cluster_cnt, sizeof(int), 1, file_ptr);
    fwrite(&this->feature_cnt, sizeof(int), 1, file_ptr);
    fwrite(this->clusters->get_arr(Cpu), sizeof(float), (this->clusters->get_row_cnt() * this->clusters->get_col_cnt()), file_ptr);

    fclose(file_ptr);
}

void KMeans::initialize_clusters(Tensor *x)
{
    this->clusters->translate(Cpu);

    std::vector<int> rand_nums;

    for (int i = 0; i < this->cluster_cnt; i++)
    {
        bool rand_num_already_added;
        int rand_num;

        do
        {
            rand_num_already_added = false;
            rand_num = rand() % x->get_row_cnt();

            for (int j = 0; j < rand_nums.size(); j++)
            {
                if (rand_nums[j] == rand_num)
                {
                    rand_num_already_added = true;
                    break;
                }
            }

        } while (rand_num_already_added);

        rand_nums.push_back(rand_num);
    }

    for (int cluster_idx = 0; cluster_idx < this->cluster_cnt; cluster_idx++)
    {
        int rand_row_idx = rand_nums[cluster_idx];
        memcpy(this->clusters->get_slice(cluster_idx * this->feature_cnt, Cpu), x->get_slice(rand_row_idx * x->get_col_cnt(), Cpu), sizeof(float) * this->feature_cnt);
    }

    this->clusters->translate(Gpu);
}

void KMeans::reset_clusters()
{
    this->clusters->set_all(0.0f);
}

float KMeans::train(Tensor *x)
{
    this->initialize_clusters(x);

    Tensor *assignments = new Tensor(x->get_row_cnt(), 1, Gpu);
    assignments->set_all(0.0f);

    Tensor *assignment_cnts = new Tensor(this->cluster_cnt, 1, Gpu);
    assignment_cnts->set_all(0.0f);

    int epoch = 1;

    float h_cost;
    float h_prv_cost = FLT_MAX;

    float *d_cost;
    hipMalloc(&d_cost, sizeof(float));
    hipMemset(d_cost, 0, sizeof(float));

    while (true)
    {
        // Assign xs to clusters:
        {
            int threads_per_block(THREADS_PER_BLOCK);
            int num_blocks((x->get_row_cnt() / threads_per_block) + 1);
            k_assign<<<num_blocks, threads_per_block>>>(x->get_arr(Gpu), assignments->get_arr(Gpu), this->clusters->get_arr(Gpu), d_cost, this->feature_cnt, this->cluster_cnt, x->get_row_cnt());
        }

        // Analyze cost:
        {
            hipMemcpy(&h_cost, d_cost, sizeof(float), hipMemcpyDeviceToHost);

            h_cost /= x->get_row_cnt();

            if (h_prv_cost <= h_cost)
            {
                break;
            }

            h_prv_cost = h_cost;
        }

        // Reset clusters prior to update:
        {
            int threads_per_block(THREADS_PER_BLOCK);
            int num_blocks(((this->cluster_cnt * this->feature_cnt) / threads_per_block) + 1);
            k_reset<<<num_blocks, threads_per_block>>>(this->clusters->get_arr(Gpu), (this->cluster_cnt * this->feature_cnt));
        }

        // Update clusters:
        {

            // Update clusters part 1:
            {
                int threads_per_block(THREADS_PER_BLOCK);
        int num_blocks((x->get_row_cnt() / threads_per_block) + 1);
        k_update_part_1<<<num_blocks, threads_per_block>>>(x->get_arr(Gpu), assignments->get_arr(Gpu), this->clusters->get_arr(Gpu),
                                                           assignment_cnts->get_arr(Gpu), this->feature_cnt, this->cluster_cnt, x->get_row_cnt());
    }

    // Update clusters part 2:
    {
        int threads_per_block(THREADS_PER_BLOCK);
        int num_blocks(((this->cluster_cnt * this->feature_cnt) / threads_per_block) + 1);
        k_update_part_2<<<num_blocks, threads_per_block>>>(this->clusters->get_arr(Gpu), assignment_cnts->get_arr(Gpu), this->cluster_cnt, this->feature_cnt);
    }
}

// Reset for next epoch:
{
    hipMemset(d_cost, 0, sizeof(float));
    assignment_cnts->set_all(0.0f);
}

epoch++;
}

    hipFree(d_cost);

    delete assignments;
    delete assignment_cnts;

    return h_cost;
}

Tensor *KMeans::predict(Tensor *x)
{
    Tensor *assignments = new Tensor(x->get_row_cnt(), 1, Gpu);

    {
        int threads_per_block(THREADS_PER_BLOCK);
        int num_blocks((x->get_row_cnt() / threads_per_block) + 1);
        k_assign<<<num_blocks, threads_per_block, sizeof(float) * this->cluster_cnt>>>(x->get_arr(Gpu), assignments->get_arr(Gpu), this->clusters->get_arr(Gpu), nullptr,
                                                                                       this->feature_cnt, this->cluster_cnt, x->get_row_cnt());
    }

    assignments->translate(Cpu);

    return assignments;
}

// KMeans static functions:

void KMeans::dump_best(Tensor *x, int cluster_cnt, int iter_cnt, const char *path)
{
    KMeans *kmeans = new KMeans(cluster_cnt, x->get_col_cnt());
    KMeans *best_kmeans = new KMeans(cluster_cnt, x->get_col_cnt());

    float cost;
    float min_cost = FLT_MAX;

    for (int i = 0; i < iter_cnt; i++)
    {
        cost = kmeans->train(x);

        if (cost < min_cost)
        {
            min_cost = cost;

            printf("LOWEST COST: %f\n", min_cost);

            best_kmeans->clusters->set_arr(kmeans->clusters->get_arr(Cpu));
        }

        kmeans->reset_clusters();
    }

    best_kmeans->dump(path);

    delete kmeans;
    delete best_kmeans;
}