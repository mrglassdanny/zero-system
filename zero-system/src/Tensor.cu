#include "Tensor.cuh"

Tensor::Tensor(int row_cnt, int col_cnt, bool gpu_flg)
{
    if (gpu_flg)
    {
        hipMalloc(&this->arr, sizeof(float) * (row_cnt * col_cnt));
    }
    else
    {
        this->arr = (float *)malloc(sizeof(float) * (row_cnt * col_cnt));
    }

    this->row_cnt = row_cnt;
    this->col_cnt = col_cnt;
    this->gpu_flg = gpu_flg;
}

Tensor::Tensor(Tensor *src)
{
    if (src->gpu_flg)
    {
        hipMalloc(&this->arr, sizeof(float) * (row_cnt * col_cnt));
        hipMemcpy(this->arr, src->arr, sizeof(float) * (row_cnt * col_cnt), hipMemcpyDeviceToDevice);
    }
    else
    {
        this->arr = (float *)malloc(sizeof(float) * (row_cnt * col_cnt));
        memcpy(this->arr, src->arr, sizeof(float) * (row_cnt * col_cnt));
    }

    this->row_cnt = src->row_cnt;
    this->col_cnt = src->col_cnt;
    this->gpu_flg = src->gpu_flg;
}

Tensor::Tensor(int row_cnt, int col_cnt, bool gpu_flg, float *cpu_arr)
{
    if (gpu_flg)
    {
        hipMalloc(&this->arr, sizeof(float) * (row_cnt * col_cnt));
        hipMemcpy(this->arr, cpu_arr, sizeof(float) * (row_cnt * col_cnt), hipMemcpyHostToDevice);
    }
    else
    {
        this->arr = (float *)malloc(sizeof(float) * (row_cnt * col_cnt));
        memcpy(this->arr, cpu_arr, sizeof(float) * (row_cnt * col_cnt));
    }

    this->row_cnt = row_cnt;
    this->col_cnt = col_cnt;
    this->gpu_flg = gpu_flg;
}

Tensor::~Tensor()
{
    if (this->gpu_flg)
    {
        hipFree(this->arr);
    }
    else
    {
        free(this->arr);
    }
}

void Tensor::translate(bool gpu_flg)
{
    if (gpu_flg)
    {
        if (!this->gpu_flg)
        {
            float *d_arr;
            hipMalloc(&d_arr, sizeof(float) * (this->row_cnt * this->col_cnt));
            hipMemcpy(d_arr, this->arr, sizeof(float) * (this->row_cnt * this->col_cnt), hipMemcpyHostToDevice);
            free(this->arr);
            this->arr = d_arr;

            this->gpu_flg = true;
        }
    }
    else
    {
        if (this->gpu_flg)
        {
            float *h_arr = (float *)malloc(sizeof(float) * (this->row_cnt * this->col_cnt));
            hipMemcpy(h_arr, this->arr, sizeof(float) * (this->row_cnt * this->col_cnt), hipMemcpyDeviceToHost);
            hipFree(this->arr);
            this->arr = h_arr;

            this->gpu_flg = false;
        }
    }
}

float Tensor::get_idx(int idx)
{
    if (this->gpu_flg)
    {
        float val;
        hipMemcpy(&val, &this->arr[idx], sizeof(float), hipMemcpyDeviceToHost);
        return val;
    }
    else
    {
        return this->arr[idx];
    }
}

float Tensor::get_rowcol(int row_idx, int col_idx)
{
    int idx = row_idx * this->col_cnt + col_idx;
    return this->get_idx(idx);
}

float *Tensor::get_arr(bool gpu_flg)
{
    this->translate(gpu_flg);
    return this->arr;
}

void Tensor::set_idx(int idx, float val)
{
    if (this->gpu_flg)
    {
        hipMemcpy(&this->arr[idx], &val, sizeof(float), hipMemcpyHostToDevice);
    }
    else
    {
        this->arr[idx] = val;
    }
}

void Tensor::set_rowcol(int row_idx, int col_idx, float val)
{
    int idx = row_idx * this->col_cnt + col_idx;
    return this->set_idx(idx, val);
}

void Tensor::set_arr(float *arr, bool gpu_flg, bool translate_flg)
{
    bool orig_gpu_flg = this->gpu_flg;

    this->translate(gpu_flg);

    if (gpu_flg)
    {
        hipMalloc(&this->arr, sizeof(float) * (row_cnt * col_cnt));
        hipMemcpy(this->arr, arr, sizeof(float) * (row_cnt * col_cnt), hipMemcpyHostToDevice);
    }
    else
    {
        this->arr = (float *)malloc(sizeof(float) * (row_cnt * col_cnt));
        memcpy(this->arr, arr, sizeof(float) * (row_cnt * col_cnt));
    }

    if (translate_flg)
    {
        this->translate(orig_gpu_flg);
    }
}

void Tensor::set_all(float val)
{
    int tot_cnt = this->row_cnt * this->col_cnt;

    bool orig_gpu_flg = this->gpu_flg;

    this->translate(false);

    for (int i = 0; i < tot_cnt; i++)
    {
        this->arr[i] = val;
    }

    if (orig_gpu_flg)
    {
        this->translate(true);
    }
}

void Tensor::set_all_rand(float upper)
{
    int tot_cnt = this->row_cnt * this->col_cnt;

    bool orig_gpu_flg = this->gpu_flg;

    this->translate(false);

    for (int i = 0; i < tot_cnt; i++)
    {
        float val = (float)rand() / ((float)RAND_MAX);
        val *= (2 * upper);
        val -= upper;
        this->arr[i] = val;
    }

    if (orig_gpu_flg)
    {
        this->translate(true);
    }
}

void Tensor::print()
{
    bool orig_gpu_flg = this->gpu_flg;

    if (orig_gpu_flg == 1)
    {
        this->translate(false);
    }

    {
        printf("[");
        for (int i = 0; i < this->row_cnt; i++)
        {

            if (i == 0)
            {
                printf(" [ ");
            }
            else
            {
                printf("  [ ");
            }

            for (int j = 0; j < this->col_cnt; j++)
            {
                if (j == this->col_cnt - 1)
                {
                    printf("%f", this->arr[i * this->col_cnt + j]);
                }
                else
                {
                    printf("%f, ", this->arr[i * this->col_cnt + j]);
                }
            }

            if (i == this->row_cnt - 1)
            {
                printf(" ] ");
            }
            else
            {
                printf(" ],\n");
            }
        }
        printf("]\n");
    }

    this->translate(orig_gpu_flg);
}