#include "hip/hip_runtime.h"
#include <iostream>

#include <zero_system/nn/nn.cuh>
#include <zero_system/nn/cnn.cuh>
#include <zero_system/cluster/kmeans.cuh>

using namespace zero::core;
using namespace zero::nn;
using namespace zero::cluster;

void nn_test()
{
	int x_col_cnt = 126;
	int y_col_cnt = 4;

	Tensor *x = new Tensor(1, x_col_cnt, Gpu);
	x->set_all(0.5f);

	Tensor *y = new Tensor(1, y_col_cnt, Gpu);
	y->set_all(0.0f);
	y->set_val(0, 1.0f);

	NN *nn = new NN(MSE, 0.01f);

	nn->add_layer(x_col_cnt, 0.2f);
	nn->add_layer(90, None, 0.5f);
	nn->add_layer(28, None, 0.35f);
	nn->add_layer(y_col_cnt, Tanh);

	nn->compile();

	nn->check_gradient(x, y, true);

	delete nn;

	delete x;
	delete y;
}

void nn_performance_test()
{
	int epoch_cnt = 100;
	int batch_size = 100;

	int x_col_cnt = 832 * 2;
	int y_col_cnt = 1;

	NN *nn = new NN(MSE, 0.01f);
	nn->add_layer(x_col_cnt);
	nn->add_layer(416, Sigmoid);
	nn->add_layer(y_col_cnt, Sigmoid);
	nn->compile();

	// -----------------------------------------------------------------

	Tensor *x = new Tensor(1, x_col_cnt, Gpu);
	x->set_all(0.5f);

	Tensor *y = new Tensor(1, y_col_cnt, Gpu);
	y->set_all(0.0f);
	y->set_val(0, 1.0f);

	printf("Starting Performance Test...\n");
	clock_t t;
	t = clock();

	for (int i = 0; i < epoch_cnt; i++)
	{
		for (int j = 0; j < batch_size; j++)
		{
			nn->feed_forward(x, false);
			nn->back_propagate(y, false);
		}
	}

	t = clock() - t;
	double time_taken = ((double)t) / CLOCKS_PER_SEC;

	printf("Performance Test Complete!\n");
	printf("Elapsed Seconds: %f\n\n", time_taken);

	delete nn;

	delete x;
	delete y;
}

void kmeans_test()
{
	Tensor *x = Tensor::from_csv("C:\\Users\\d0g0825\\Desktop\\temp\\kmeans\\data.csv");

	KMeans::dump_best(x, 3, 10000, "C:\\Users\\d0g0825\\Desktop\\temp\\kmeans\\model.km");

	KMeans *km = new KMeans("C:\\Users\\d0g0825\\Desktop\\temp\\kmeans\\model.km");

	km->print();

	delete km;

	delete x;
}

void cnn_test()
{
	int x_col_cnt = 2 * 16 * 16;
	int y_col_cnt = 4;

	Tensor *x = new Tensor(1, x_col_cnt, Gpu);
	x->set_all_rand(1.0f);

	Tensor *y = new Tensor(1, y_col_cnt, Gpu);
	y->set_all(0.0f);
	y->set_val(0, 1.0f);

	CNN *cnn = new CNN(MSE, 0.001f);
	cnn->add_layer(2, 16, 16, 4, 4, 4, Sigmoid);
	cnn->add_layer(3, 2, 2, None);
	cnn->add_layer(None);
	cnn->compile();

	cnn->fully_connected()->add_layer(24, Sigmoid);
	cnn->fully_connected()->add_layer(y_col_cnt, Sigmoid);
	cnn->fully_connected()->compile();

	cnn->check_gradient(x, y, true);

	delete cnn;

	delete x;
	delete y;
}

int main(int argc, char **argv)
{
	srand(time(NULL));

	//nn_test();

	//nn_performance_test();

	//kmeans_test();

	cnn_test();

	return 0;
}