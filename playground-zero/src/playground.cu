#include "hip/hip_runtime.h"
#include <iostream>

#include <zero_system/nn/nn.cuh>
#include <zero_system/cluster/kmeans.cuh>

using namespace zero::core;
using namespace zero::nn;
using namespace zero::cluster;

void nn_test()
{
	int x_col_cnt = 16;
	int y_col_cnt = 4;

	Tensor *x = new Tensor(1, x_col_cnt, Gpu);
	x->set_all(0.5f);

	Tensor *y = new Tensor(1, y_col_cnt, Gpu);
	y->set_all(0.0f);
	y->set_idx(0, 1.0f);

	NN *nn = new NN(MSE, 0.01f);

	nn->add_layer(x_col_cnt, 0.0f);
	nn->add_layer(10, Tanh, 0.5f);
	nn->add_layer(6, Sigmoid, 0.0f);
	nn->add_layer(y_col_cnt, Sigmoid);

	nn->compile();

	nn->check_gradient(x, y, true);

	delete nn;

	delete x;
	delete y;
}

void kmeans_test()
{
	Tensor *x = Tensor::from_csv("C:\\Users\\d0g0825\\Desktop\\temp\\kmeans\\data.csv");

	KMeans::dump_best(x, 3, 10000, "C:\\Users\\d0g0825\\Desktop\\temp\\kmeans\\model.km");

	KMeans *km = new KMeans("C:\\Users\\d0g0825\\Desktop\\temp\\kmeans\\model.km");

	km->print();

	delete km;

	delete x;
}

void nn_performance_test()
{
	int epoch_cnt = 100;
	int batch_size = 100;

	int x_col_cnt = 832 * 2;
	int y_col_cnt = 1;

	NN *nn = new NN(MSE, 0.01f);
	nn->add_layer(x_col_cnt);
	nn->add_layer(416, Sigmoid);
	nn->add_layer(y_col_cnt, Sigmoid);
	nn->compile();

	// -----------------------------------------------------------------

	Tensor *x = new Tensor(1, x_col_cnt, Gpu);
	x->set_all(0.5f);

	Tensor *y = new Tensor(1, y_col_cnt, Gpu);
	y->set_all(0.0f);
	y->set_idx(0, 1.0f);

	printf("Starting Performance Test...\n");
	clock_t t;
	t = clock();

	for (int i = 0; i < epoch_cnt; i++)
	{
		for (int j = 0; j < batch_size; j++)
		{
			nn->feed_forward(x, false);
			nn->back_propagate(y);
		}
	}

	t = clock() - t;
	double time_taken = ((double)t) / CLOCKS_PER_SEC;

	printf("Performance Test Complete!\n");
	printf("Elapsed Seconds: %f\n\n", time_taken);

	delete nn;

	delete x;
	delete y;
}

int main(int argc, char **argv)
{
	srand(time(NULL));

	nn_test();

	//kmeans_test();

	//nn_performance_test();

	return 0;
}