#include "hip/hip_runtime.h"
#include <iostream>

#include <zero_system/nn/nn.cuh>
#include <zero_system/cluster/kmeans.cuh>

using namespace zero::core;
using namespace zero::nn;
using namespace zero::cluster;

void nn_test()
{
	int x_col_cnt = 26;
	int y_col_cnt = 1;

	Tensor *x = new Tensor(1, x_col_cnt, Gpu);
	x->set_all(0.5f);

	Tensor *y = new Tensor(1, y_col_cnt, Gpu);
	y->set_all(0.0f);
	y->set_idx(0, 1.0f);

	std::vector<int> layer_config = {x_col_cnt, 16, 8, 6, 4, y_col_cnt};
	NN *nn = new NN(layer_config, Tanh, Sigmoid, MSE, 0.01f);

	nn->check_gradient(x, y, true);

	delete nn;

	delete x;
	delete y;
}

void kmeans_test()
{
	Tensor *x = Tensor::from_csv("C:\\Users\\d0g0825\\Desktop\\temp\\kmeans\\data.csv");

	KMeans::dump_best(x, 3, 10000, "C:\\Users\\d0g0825\\Desktop\\temp\\kmeans\\model.km");

	KMeans *km = new KMeans("C:\\Users\\d0g0825\\Desktop\\temp\\kmeans\\model.km");

	km->print();

	delete km;

	delete x;
}

void nn_performance_test()
{
	int epoch_cnt = 100;
	int batch_size = 100;

	int x_col_cnt = 832 * 2;
	int y_col_cnt = 1;

	std::vector<int> layer_config = {x_col_cnt, 2048, 2048, 1024, 1024, 256, 64, 16, y_col_cnt};

	// -----------------------------------------------------------------

	Tensor *x = new Tensor(1, x_col_cnt, Gpu);
	x->set_all(0.5f);

	Tensor *y = new Tensor(1, y_col_cnt, Gpu);
	y->set_all(0.0f);
	y->set_idx(0, 1.0f);

	NN *nn = new NN(layer_config, ReLU, ReLU, MSE, 0.01f);

	printf("Starting Performance Test...\n");
	clock_t t;
	t = clock();

	for (int i = 0; i < epoch_cnt; i++)
	{
		for (int j = 0; j < batch_size; j++)
		{
			nn->feed_forward(x, 0.0f);
			nn->back_propagate(y);
		}
	}

	t = clock() - t;
	double time_taken = ((double)t) / CLOCKS_PER_SEC;

	printf("Performance Test Complete!\n");
	printf("Elapsed Seconds: %f\n\n", time_taken);

	delete nn;

	delete x;
	delete y;
}

int main(int argc, char **argv)
{
	srand(time(NULL));

	nn_test();

	//kmeans_test();

	//nn_performance_test();

	return 0;
}