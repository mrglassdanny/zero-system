#include "hip/hip_runtime.h"

#include <zero_system/mod.cuh>

#define LOC_MODEL_OUTPUT_N_CNT 8

std::vector<int> get_output_shape()
{
    std::vector<int> n_shape{1};
    return n_shape;
}

void forward(Tensor *n, Tensor *nxt_n, bool train_flg)
{
    // t = aq + c + v

    float a = n->get_val(0);
    float c = n->get_val(1);
    float q = n->get_val(2);
    float v = 0.0f;

    int src_loc_beg_idx = 3;
    int dst_loc_beg_idx = src_loc_beg_idx + LOC_MODEL_OUTPUT_N_CNT;

    for (int i = 0; i < LOC_MODEL_OUTPUT_N_CNT; i++)
    {
        float loc_diff = n->get_val(src_loc_beg_idx + i) - n->get_val(dst_loc_beg_idx + i);
        v += (loc_diff * loc_diff);
    }

    v = sqrt(v);

    float t = a * q + c + v;

    nxt_n->set_val(0, t);
}

void forward2(Tensor *n, Tensor *nxt_n, bool train_flg)
{
    // t = a + v

    float a = n->get_val(0);
    float v = 0.0f;

    int src_loc_beg_idx = 1;
    int dst_loc_beg_idx = src_loc_beg_idx + LOC_MODEL_OUTPUT_N_CNT;

    for (int i = 0; i < LOC_MODEL_OUTPUT_N_CNT; i++)
    {
        float loc_diff = n->get_val(src_loc_beg_idx + i) - n->get_val(dst_loc_beg_idx + i);
        v += (loc_diff * loc_diff);
    }

    v = sqrt(v);

    float t = a + v;

    nxt_n->set_val(0, t);
}

Tensor *backward(Tensor *n, Tensor *dc)
{
    // t = aq + c + v

    Tensor *nxt_dc = new Tensor(n->get_device(), n->get_shape());

    float dc_val = dc->get_val(0);

    nxt_dc->set_val(0, dc_val * n->get_val(2));
    nxt_dc->set_val(1, dc_val * 1.0f);
    nxt_dc->set_val(2, dc_val * n->get_val(0));

    int src_loc_beg_idx = 3;
    int dst_loc_beg_idx = src_loc_beg_idx + LOC_MODEL_OUTPUT_N_CNT;

    float v = 0.0f;

    for (int i = 0; i < LOC_MODEL_OUTPUT_N_CNT; i++)
    {
        float loc_diff = n->get_val(src_loc_beg_idx + i) - n->get_val(dst_loc_beg_idx + i);
        v += (loc_diff * loc_diff);
    }

    v = sqrt(v);

    for (int i = 0; i < LOC_MODEL_OUTPUT_N_CNT; i++)
    {
        if (v == 0.0f)
        {
            nxt_dc->set_val(src_loc_beg_idx + i, 0.0f);
            nxt_dc->set_val(dst_loc_beg_idx + i, 0.0f);
        }
        else
        {
            float dv = 1.0f / (2.0f * v);

            nxt_dc->set_val(src_loc_beg_idx + i, dc_val * dv * (2.0f * (n->get_val(src_loc_beg_idx + i) - n->get_val(dst_loc_beg_idx + i))));
            nxt_dc->set_val(dst_loc_beg_idx + i, dc_val * dv * (-2.0f * (n->get_val(src_loc_beg_idx + i) - n->get_val(dst_loc_beg_idx + i))));
        }
    }

    return nxt_dc;
}

Tensor *backward2(Tensor *n, Tensor *dc)
{
    // t = a + v

    Tensor *nxt_dc = new Tensor(n->get_device(), n->get_shape());

    float dc_val = dc->get_val(0);

    nxt_dc->set_val(0, dc_val * 1.0f);

    int src_loc_beg_idx = 1;
    int dst_loc_beg_idx = src_loc_beg_idx + LOC_MODEL_OUTPUT_N_CNT;

    float v = 0.0f;

    for (int i = 0; i < LOC_MODEL_OUTPUT_N_CNT; i++)
    {
        float loc_diff = n->get_val(src_loc_beg_idx + i) - n->get_val(dst_loc_beg_idx + i);
        v += (loc_diff * loc_diff);
    }

    v = sqrt(v);

    for (int i = 0; i < LOC_MODEL_OUTPUT_N_CNT; i++)
    {
        if (v == 0.0f)
        {
            nxt_dc->set_val(src_loc_beg_idx + i, 0.0f);
            nxt_dc->set_val(dst_loc_beg_idx + i, 0.0f);
        }
        else
        {
            float dv = 1.0f / (2.0f * v);

            nxt_dc->set_val(src_loc_beg_idx + i, dc_val * dv * (2.0f * (n->get_val(src_loc_beg_idx + i) - n->get_val(dst_loc_beg_idx + i))));
            nxt_dc->set_val(dst_loc_beg_idx + i, dc_val * dv * (-2.0f * (n->get_val(src_loc_beg_idx + i) - n->get_val(dst_loc_beg_idx + i))));
        }
    }

    return nxt_dc;
}

void upd_rslt_fn(Tensor *p, Tensor *y, int *cnt)
{
    float y_val = y->get_val(0);
    float p_val = p->get_val(0);

    float lower = y_val < p_val ? y_val : p_val;
    float upper = y_val < p_val ? p_val : y_val;

    float prcnt = 1.0f - (lower / upper);

    if (prcnt <= 0.20f)
    {
        (*cnt)++;
    }
}

std::vector<float> loc_encode_fn(const char *loc_name, int dim_cnt)
{
    char delims[] = {'-'};
    char numerics[] = {'0', '1', '2', '3', '4', '5', '6', '7', '8', '9'};

    StackBuffer buf;

    std::vector<float> parsed_loc;

    int loc_name_len = strlen(loc_name);

    bool delim_flg = false;
    bool numeric_flg = false;
    bool alpha_flg = false;

    for (int i = 0; i < loc_name_len; i++)
    {
        char c = loc_name[i];

        for (int j = 0; j < sizeof(delims); j++)
        {
            if (c == delims[j])
            {
                if (buf.get_size() > 0)
                {
                    parsed_loc.push_back(atof(buf.get()));
                }

                buf.clear();

                delim_flg = true;
                numeric_flg = false;
                alpha_flg = false;

                break;
            }
        }

        if (!delim_flg)
        {
            numeric_flg = false;

            for (int j = 0; j < sizeof(numerics); j++)
            {
                if (c == numerics[j])
                {
                    numeric_flg = true;

                    if (alpha_flg)
                    {
                        if (buf.get_size() > 0)
                        {
                            parsed_loc.push_back(atof(buf.get()));
                        }

                        buf.clear();
                        buf.append(c);
                    }
                    else
                    {
                        buf.append(c);
                    }

                    alpha_flg = false;

                    break;
                }
            }
        }

        if (!delim_flg && !numeric_flg)
        {
            if (alpha_flg)
            {
                float buf_num = 0.0f;

                if (buf.get_size() > 0)
                {
                    buf_num = atof(buf.get());
                }

                buf.clear();
                buf_num += (int)c;
                buf.append(buf_num);
            }
            else
            {
                if (buf.get_size() > 0)
                {
                    parsed_loc.push_back(atof(buf.get()));
                }

                buf.clear();
                buf.append((int)c);
            }

            alpha_flg = true;
        }
    }

    if (buf.get_size() > 0)
    {
        parsed_loc.push_back(atof(buf.get()));
    }

    if (parsed_loc.size() < dim_cnt)
    {
        for (int i = parsed_loc.size(); i < dim_cnt; i++)
        {
            parsed_loc.push_back(0.0f);
        }
    }

    return parsed_loc;
}

void fit(Table *xs_tbl, Table *ys_tbl, Supervisor *sup)
{
    Model *lm = new Model(0.1f);

    Model *variable_act_model = new Model();
    variable_act_model->dense(xs_tbl->get_last_column_idx("typ") - xs_tbl->get_column_idx("actcod") + 1, 32);
    variable_act_model->activation(Tanh);
    variable_act_model->dense(8);
    variable_act_model->activation(Tanh);
    variable_act_model->dense(1);
    variable_act_model->activation(Tanh);

    // Model *constant_act_model = new Model();
    // constant_act_model->copy(variable_act_model);

    Model *src_loc_model = new Model();
    src_loc_model->dense(3, 32);
    src_loc_model->activation(Tanh);
    src_loc_model->dense(16);
    src_loc_model->activation(Tanh);
    src_loc_model->dense(LOC_MODEL_OUTPUT_N_CNT);
    src_loc_model->activation(Tanh);

    Model *dst_loc_model = new Model();
    dst_loc_model->copy(src_loc_model);
    dst_loc_model->share_parameters(src_loc_model);

    lm->child(variable_act_model, Range{xs_tbl->get_column_idx("actcod"), xs_tbl->get_last_column_idx("typ")});
    // lm->child(constant_act_model, Range{xs_tbl->get_column_idx("constant_actcod"), xs_tbl->get_last_column_idx("constant_typ")});
    lm->child(src_loc_model, xs_tbl->get_column_range("fr_loc"));
    lm->child(dst_loc_model, xs_tbl->get_column_range("to_loc"));

    lm->custom(lm->calc_adjusted_input_shape(xs_tbl->get_column_cnt()),
               get_output_shape, forward2, backward2);
    lm->activation(Tanh);

    lm->fit(sup, 25, 30, "temp/train.csv", upd_rslt_fn);

    Batch *test_batch = sup->create_batch();
    lm->test(test_batch, upd_rslt_fn).print();
    delete test_batch;

    lm->save("temp/lm.model");
    variable_act_model->save("temp/vact.model");
    // constant_act_model->save("temp/cact.model");
    src_loc_model->save("temp/loc.model");

    delete lm;
    delete variable_act_model;
    // delete constant_act_model;
    delete src_loc_model;
    delete dst_loc_model;
}

// Using this fn will cause memory leak for child models!
Model *load_lm()
{
    Model *lm = new Model();
    lm->load("temp/lm.model");

    Model *variable_act_model = new Model();
    variable_act_model->load("temp/vact.model");

    // Model *constant_act_model = new Model();
    // constant_act_model->load("temp/cact.model");

    Model *src_loc_model = new Model();
    src_loc_model->load("temp/loc.model");

    Model *dst_loc_model = new Model();
    dst_loc_model->load("temp/loc.model");
    dst_loc_model->share_parameters(src_loc_model);

    lm->child(variable_act_model);
    // lm->child(constant_act_model);
    lm->child(src_loc_model);
    lm->child(dst_loc_model);

    ((CustomLayer *)lm->get_layers()[0])->set_callbacks(get_output_shape, forward2, backward2);

    return lm;
}

void test(Supervisor *sup, Column *pred_col)
{
    Model *lm = load_lm();

    Batch *test_batch = sup->create_batch();
    lm->test(test_batch, upd_rslt_fn).print();

    for (int i = 0; i < test_batch->get_size(); i++)
    {
        Tensor *pred = lm->forward(test_batch->get_x(i), false);
        pred_col->set_val(i, pred->get_val(0));
        delete pred;
    }

    delete test_batch;

    delete lm;
}

void grad_check(Table *xs_tbl, Table *ys_tbl, Supervisor *sup)
{
    Model *lm = load_lm();

    Batch *grad_check_batch = sup->create_batch();

    lm->grad_check(grad_check_batch->get_x(1), grad_check_batch->get_y(1), true);

    delete grad_check_batch;

    delete lm;
}

int main(int argc, char **argv)
{
    ZERO();

    // Data setup:

    Table *xs_tbl = Table::fr_csv("data/palmov-test.csv");
    Table *ys_tbl = xs_tbl->split("elapsed_secs");

    delete xs_tbl->remove_column("cas_per_lyr");
    delete xs_tbl->remove_column("lyr_per_pal");
    delete xs_tbl->remove_column("cas_len");
    delete xs_tbl->remove_column("cas_wid");
    delete xs_tbl->remove_column("cas_hgt");
    delete xs_tbl->remove_column("cas_wgt");
    delete xs_tbl->remove_column("cas_qty");

    delete xs_tbl->remove_column("pal_qty");

    // Column *constant_actcod_col = new Column("constant_actcod", *xs_tbl->get_column("actcod"));
    // Column *constant_typ_col = new Column("constant_typ", *xs_tbl->get_column("typ"));

    // xs_tbl->add_column(constant_actcod_col, "typ");
    // xs_tbl->add_column(constant_typ_col, "constant_actcod");

    Column *actcod_col = new Column(*xs_tbl->get_column("actcod"));
    Column *typ_col = new Column(*xs_tbl->get_column("typ"));
    Column *fr_loc_col = new Column(*xs_tbl->get_column("fr_loc"));
    Column *to_loc_col = new Column(*xs_tbl->get_column("to_loc"));

    xs_tbl->encode_onehot("actcod");
    xs_tbl->encode_onehot("typ");
    // xs_tbl->encode_onehot("constant_actcod");
    // xs_tbl->encode_onehot("constant_typ");
    xs_tbl->encode_custom("fr_loc", 3, loc_encode_fn);
    xs_tbl->encode_custom("to_loc", 3, loc_encode_fn);

    xs_tbl->scale_down();
    ys_tbl->scale_down();

    Supervisor *sup;
    {
        Tensor *xs = Table::to_tensor(xs_tbl);
        Tensor *ys = Table::to_tensor(ys_tbl);

        Tensor::to_file("temp/xs.tr", xs);
        Tensor::to_file("temp/ys.tr", ys);

        std::vector<int> x_shape{xs->get_shape()[1]};
        sup = new Supervisor("temp/xs.tr", "temp/ys.tr", x_shape, 0);

        delete xs;
        delete ys;
    }

    // Fit:
    {
        fit(xs_tbl, ys_tbl, sup);
    }

    // Test:
    {
        Column *y_col = ys_tbl->get_column("elapsed_secs");
        Column *pred_col = new Column("pred", true, xs_tbl->get_row_cnt());

        xs_tbl->clear();

        xs_tbl->add_column(actcod_col);
        xs_tbl->add_column(typ_col);
        xs_tbl->add_column(fr_loc_col);
        xs_tbl->add_column(to_loc_col);
        xs_tbl->add_column(y_col);
        xs_tbl->add_column(pred_col);

        test(sup, pred_col);

        Table::to_csv("temp/preds.csv", xs_tbl);
    }

    // Grad Check:
    {
        // grad_check(xs_tbl, ys_tbl, sup);
    }

    // Cleanup:

    delete sup;

    return 0;
}