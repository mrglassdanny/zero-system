#include "hip/hip_runtime.h"

#include <zero_system/mod.cuh>

#define ACTCODTYP_EMBG_DIM_CNT 4
#define LOC_EMBG_DIM_CNT 12

std::vector<int> get_output_shape()
{
    std::vector<int> n_shape{ACTCODTYP_EMBG_DIM_CNT + LOC_EMBG_DIM_CNT};
    return n_shape;
}

std::vector<int> get_output_shape2()
{
    std::vector<int> n_shape{2};
    return n_shape;
}

std::vector<int> get_output_shape3()
{
    std::vector<int> n_shape{ACTCODTYP_EMBG_DIM_CNT + 1};
    return n_shape;
}

void forward(Tensor *n, Tensor *nxt_n, bool train_flg)
{
    for (int i = 0; i < ACTCODTYP_EMBG_DIM_CNT; i++)
    {
        nxt_n->set_val(i, n->get_val(i));
    }

    int src_loc_beg_idx = ACTCODTYP_EMBG_DIM_CNT;
    int dst_loc_beg_idx = src_loc_beg_idx + LOC_EMBG_DIM_CNT;

    for (int i = 0; i < LOC_EMBG_DIM_CNT; i++)
    {
        float loc_diff = n->get_val(src_loc_beg_idx + i) - n->get_val(dst_loc_beg_idx + i);
        nxt_n->set_val(i + ACTCODTYP_EMBG_DIM_CNT, loc_diff);
    }
}

void forward2(Tensor *n, Tensor *nxt_n, bool train_flg)
{
    float a = n->get_val(0);
    float t = 0.0f;

    int src_loc_beg_idx = 1;
    int dst_loc_beg_idx = src_loc_beg_idx + LOC_EMBG_DIM_CNT;

    for (int i = 0; i < LOC_EMBG_DIM_CNT; i++)
    {
        float loc_diff = n->get_val(src_loc_beg_idx + i) - n->get_val(dst_loc_beg_idx + i);
        t += (loc_diff * loc_diff);
    }

    t = sqrt(t);

    nxt_n->set_val(0, a);
    nxt_n->set_val(1, t);
}

void forward3(Tensor *n, Tensor *nxt_n, bool train_flg)
{
    for (int i = 0; i < ACTCODTYP_EMBG_DIM_CNT; i++)
    {
        nxt_n->set_val(i, n->get_val(i));
    }

    int src_loc_beg_idx = ACTCODTYP_EMBG_DIM_CNT;
    int dst_loc_beg_idx = src_loc_beg_idx + LOC_EMBG_DIM_CNT;

    float t = 0.0f;

    for (int i = 0; i < LOC_EMBG_DIM_CNT; i++)
    {
        float loc_diff = n->get_val(src_loc_beg_idx + i) - n->get_val(dst_loc_beg_idx + i);
        t += (loc_diff * loc_diff);
    }

    t = sqrt(t);

    nxt_n->set_val(ACTCODTYP_EMBG_DIM_CNT, t);
}

Tensor *backward(Tensor *n, Tensor *dc)
{
    Tensor *nxt_dc = new Tensor(n->get_device(), n->get_shape());

    for (int i = 0; i < ACTCODTYP_EMBG_DIM_CNT; i++)
    {
        nxt_dc->set_val(i, dc->get_val(i));
    }

    int src_loc_beg_idx = ACTCODTYP_EMBG_DIM_CNT;
    int dst_loc_beg_idx = src_loc_beg_idx + LOC_EMBG_DIM_CNT;

    for (int i = 0; i < LOC_EMBG_DIM_CNT; i++)
    {
        nxt_dc->set_val(src_loc_beg_idx + i, dc->get_val(i + ACTCODTYP_EMBG_DIM_CNT) * 1.0f);
        nxt_dc->set_val(dst_loc_beg_idx + i, dc->get_val(i + ACTCODTYP_EMBG_DIM_CNT) * -1.0f);
    }

    return nxt_dc;
}

Tensor *backward2(Tensor *n, Tensor *dc)
{
    Tensor *nxt_dc = new Tensor(n->get_device(), n->get_shape());

    nxt_dc->set_val(0, dc->get_val(0) * 1.0f);

    int src_loc_beg_idx = 1;
    int dst_loc_beg_idx = src_loc_beg_idx + LOC_EMBG_DIM_CNT;

    float t = 0.0f;

    for (int i = 0; i < LOC_EMBG_DIM_CNT; i++)
    {
        float loc_diff = n->get_val(src_loc_beg_idx + i) - n->get_val(dst_loc_beg_idx + i);
        t += (loc_diff * loc_diff);
    }

    t = sqrt(t);

    for (int i = 0; i < LOC_EMBG_DIM_CNT; i++)
    {
        float dc_val = dc->get_val(1);

        if (t == 0.0f)
        {
            nxt_dc->set_val(src_loc_beg_idx + i, 0.0f);
            nxt_dc->set_val(dst_loc_beg_idx + i, 0.0f);
        }
        else
        {
            float dv = 1.0f / (2.0f * t);

            nxt_dc->set_val(src_loc_beg_idx + i, dc_val * dv * (2.0f * (n->get_val(src_loc_beg_idx + i) - n->get_val(dst_loc_beg_idx + i))));
            nxt_dc->set_val(dst_loc_beg_idx + i, dc_val * dv * (-2.0f * (n->get_val(src_loc_beg_idx + i) - n->get_val(dst_loc_beg_idx + i))));
        }
    }

    return nxt_dc;
}

Tensor *backward3(Tensor *n, Tensor *dc)
{
    Tensor *nxt_dc = new Tensor(n->get_device(), n->get_shape());

    for (int i = 0; i < ACTCODTYP_EMBG_DIM_CNT; i++)
    {
        nxt_dc->set_val(i, dc->get_val(i));
    }

    int src_loc_beg_idx = ACTCODTYP_EMBG_DIM_CNT;
    int dst_loc_beg_idx = src_loc_beg_idx + LOC_EMBG_DIM_CNT;

    float t = 0.0f;

    for (int i = 0; i < LOC_EMBG_DIM_CNT; i++)
    {
        float loc_diff = n->get_val(src_loc_beg_idx + i) - n->get_val(dst_loc_beg_idx + i);
        t += (loc_diff * loc_diff);
    }

    t = sqrt(t);

    for (int i = 0; i < LOC_EMBG_DIM_CNT; i++)
    {
        float dc_val = dc->get_val(ACTCODTYP_EMBG_DIM_CNT);

        if (t == 0.0f)
        {
            nxt_dc->set_val(src_loc_beg_idx + i, 0.0f);
            nxt_dc->set_val(dst_loc_beg_idx + i, 0.0f);
        }
        else
        {
            float dv = 1.0f / (2.0f * t);

            nxt_dc->set_val(src_loc_beg_idx + i, dc_val * dv * (2.0f * (n->get_val(src_loc_beg_idx + i) - n->get_val(dst_loc_beg_idx + i))));
            nxt_dc->set_val(dst_loc_beg_idx + i, dc_val * dv * (-2.0f * (n->get_val(src_loc_beg_idx + i) - n->get_val(dst_loc_beg_idx + i))));
        }
    }

    return nxt_dc;
}

void upd_rslt_fn(Tensor *p, Tensor *y, int *cnt)
{
    float y_val = y->get_val(0);
    float p_val = p->get_val(0);

    float lower = y_val < p_val ? y_val : p_val;
    float upper = y_val < p_val ? p_val : y_val;

    float prcnt = 1.0f - (lower / upper);

    if (prcnt <= 0.20f)
    {
        (*cnt)++;
    }
}

std::vector<float> loc_encode_fn(const char *loc_name, int dim_cnt)
{
    char delims[] = {'-'};
    char numerics[] = {'0', '1', '2', '3', '4', '5', '6', '7', '8', '9'};

    StackBuffer buf;

    std::vector<float> parsed_loc;

    int loc_name_len = strlen(loc_name);

    bool delim_flg = false;
    bool numeric_flg = false;
    bool alpha_flg = false;

    for (int i = 0; i < loc_name_len; i++)
    {
        char c = loc_name[i];

        for (int j = 0; j < sizeof(delims); j++)
        {
            if (c == delims[j])
            {
                if (buf.get_size() > 0)
                {
                    parsed_loc.push_back(atof(buf.get()));
                }

                buf.clear();

                delim_flg = true;
                numeric_flg = false;
                alpha_flg = false;

                break;
            }
        }

        if (!delim_flg)
        {
            numeric_flg = false;

            for (int j = 0; j < sizeof(numerics); j++)
            {
                if (c == numerics[j])
                {
                    numeric_flg = true;

                    if (alpha_flg)
                    {
                        if (buf.get_size() > 0)
                        {
                            parsed_loc.push_back(atof(buf.get()));
                        }

                        buf.clear();
                        buf.append(c);
                    }
                    else
                    {
                        buf.append(c);
                    }

                    alpha_flg = false;

                    break;
                }
            }
        }

        if (!delim_flg && !numeric_flg)
        {
            if (alpha_flg)
            {
                float buf_num = 0.0f;

                if (buf.get_size() > 0)
                {
                    buf_num = atof(buf.get());
                }

                buf.clear();
                buf_num += (int)c;
                buf.append(buf_num);
            }
            else
            {
                if (buf.get_size() > 0)
                {
                    parsed_loc.push_back(atof(buf.get()));
                }

                buf.clear();
                buf.append((int)c);
            }

            alpha_flg = true;
        }
    }

    if (buf.get_size() > 0)
    {
        parsed_loc.push_back(atof(buf.get()));
    }

    if (parsed_loc.size() < dim_cnt)
    {
        for (int i = parsed_loc.size(); i < dim_cnt; i++)
        {
            parsed_loc.push_back(0.0f);
        }
    }

    return parsed_loc;
}

int main(int argc, char **argv)
{
    ZERO();

    // Data setup:

    Table *xs_tbl = Table::fr_csv("data/palmov_data-test.csv");
    Table *ys_tbl = xs_tbl->split("elapsed_secs");

    Table *actcodtyps_tbl = Table::fr_csv("data/actcodtyps.csv");
    std::map<std::string, int> *actcodtyp_map = actcodtyps_tbl->get_column(0)->to_ordinal_map();

    Table *locs_tbl = Table::fr_csv("data/locs.csv");
    std::map<std::string, int> *loc_map = locs_tbl->get_column(0)->to_ordinal_map();

    Column *actcod_col = new Column(*xs_tbl->get_column("actcod"));
    Column *typ_col = new Column(*xs_tbl->get_column("typ"));
    Column *fr_loc_col = new Column(*xs_tbl->get_column("fr_loc"));
    Column *to_loc_col = new Column(*xs_tbl->get_column("to_loc"));

    delete xs_tbl->remove_column("actcod");
    delete xs_tbl->remove_column("typ");
    // delete xs_tbl->remove_column("fr_loc");
    // delete xs_tbl->remove_column("to_loc");
    delete xs_tbl->remove_column("fx");
    delete xs_tbl->remove_column("fy");
    delete xs_tbl->remove_column("tx");
    delete xs_tbl->remove_column("ty");
    delete xs_tbl->remove_column("trvl");

    xs_tbl->encode_ordinal("actcodtyp", actcodtyp_map);
    // xs_tbl->encode_onehot("actcodtyp", actcodtyp_map);
    xs_tbl->encode_ordinal("fr_loc", loc_map);
    xs_tbl->encode_ordinal("to_loc", loc_map);

    // xs_tbl->get_column("trvl")->scale_down();
    ys_tbl->scale_down();

    Supervisor *sup;
    {
        Tensor *xs = Table::to_tensor(xs_tbl);
        Tensor *ys = Table::to_tensor(ys_tbl);

        Tensor::to_file("temp/xs.tr", xs);
        Tensor::to_file("temp/ys.tr", ys);

        std::vector<int> x_shape{xs->get_shape()[1]};
        sup = new Supervisor("temp/xs.tr", "temp/ys.tr", x_shape, 0);

        delete xs;
        delete ys;
    }

    // ===================================================================================================

    // Models:

    Model *lm = new Model(0.1f);

    // Model 1: 25%
    {
        // Model *actcodtyp_model = new Model();
        // actcodtyp_model->embedding((int)actcodtyps_tbl->get_column(0)->row_cnt, ACTCODTYP_EMBG_DIM_CNT);

        // lm->child(actcodtyp_model, xs_tbl->get_column_range("actcodtyp"));

        // lm->dense(lm->calc_adjusted_input_shape(xs_tbl->get_column_cnt()), 128);
        // lm->activation(ReLU);
        // lm->dense(32);
        // lm->activation(ReLU);
        // lm->dense(1);
        // lm->activation(ReLU);
    }

    // Model 2: 26%
    {
        // Model *actcodtyp_model = new Model();
        // actcodtyp_model->embedding((int)actcodtyps_tbl->get_column(0)->row_cnt, ACTCODTYP_EMBG_DIM_CNT);

        // Model *src_loc_model = new Model();
        // src_loc_model->embedding((int)locs_tbl->get_column(0)->row_cnt, LOC_EMBG_DIM_CNT);

        // Model *dst_loc_model = new Model();
        // dst_loc_model->copy(src_loc_model);
        // dst_loc_model->share_parameters(src_loc_model);

        // lm->child(actcodtyp_model, xs_tbl->get_column_range("actcodtyp"));
        // lm->child(src_loc_model, xs_tbl->get_column_range("fr_loc"));
        // lm->child(dst_loc_model, xs_tbl->get_column_range("to_loc"));

        // lm->dense(lm->calc_adjusted_input_shape(xs_tbl->get_column_cnt()), 256);
        // lm->activation(ReLU);
        // lm->dense(64);
        // lm->activation(ReLU);
        // lm->dense(1);
        // lm->activation(ReLU);
    }

    // Model 3: 26%
    {
        // Model *actcodtyp_model = new Model();
        // actcodtyp_model->embedding((int)actcodtyps_tbl->get_column(0)->row_cnt, ACTCODTYP_EMBG_DIM_CNT);
        // actcodtyp_model->dense(64);
        // actcodtyp_model->activation(Tanh);
        // actcodtyp_model->dense(1);

        // Model *src_loc_model = new Model();
        // src_loc_model->embedding((int)locs_tbl->get_column(0)->row_cnt, LOC_EMBG_DIM_CNT);

        // Model *dst_loc_model = new Model();
        // dst_loc_model->copy(src_loc_model);
        // dst_loc_model->share_parameters(src_loc_model);

        // lm->child(actcodtyp_model, xs_tbl->get_column_range("actcodtyp"));
        // lm->child(src_loc_model, xs_tbl->get_column_range("fr_loc"));
        // lm->child(dst_loc_model, xs_tbl->get_column_range("to_loc"));

        // lm->custom(lm->calc_adjusted_input_shape(xs_tbl->get_column_cnt()),
        //            get_output_shape2, forward2, backward2);
        // lm->activation(Tanh);
        // lm->dense(32);
        // lm->activation(Tanh);
        // lm->dense(8);
        // lm->activation(Tanh);
        // lm->dense(1);
    }

    // Model 4: 32%
    {
        Model *actcodtyp_model = new Model();
        actcodtyp_model->embedding((int)actcodtyps_tbl->get_column(0)->row_cnt, ACTCODTYP_EMBG_DIM_CNT);

        Model *src_loc_model = new Model();
        src_loc_model->embedding((int)locs_tbl->get_column(0)->row_cnt, LOC_EMBG_DIM_CNT);

        Model *dst_loc_model = new Model();
        dst_loc_model->copy(src_loc_model);
        dst_loc_model->share_parameters(src_loc_model);

        lm->child(actcodtyp_model, xs_tbl->get_column_range("actcodtyp"));
        lm->child(src_loc_model, xs_tbl->get_column_range("fr_loc"));
        lm->child(dst_loc_model, xs_tbl->get_column_range("to_loc"));

        lm->custom(lm->calc_adjusted_input_shape(xs_tbl->get_column_cnt()),
                   get_output_shape3, forward3, backward3);
        lm->activation(Tanh);
        lm->dense(32);
        lm->activation(Tanh);
        lm->dense(8);
        lm->activation(Tanh);
        lm->dense(1);
    }

    // ===================================================================================================

    // Fit:
    {
        lm->fit(sup, 25, 4, "temp/train.csv", upd_rslt_fn);

        Batch *test_batch = sup->create_batch();
        lm->test(test_batch, upd_rslt_fn).print();
        delete test_batch;
    }

    // ===================================================================================================

    // Test:
    {
        Column *y_col = ys_tbl->get_column("elapsed_secs");
        Column *pred_col = new Column("pred", true, xs_tbl->get_row_cnt());

        xs_tbl->clear();

        xs_tbl->add_column(actcod_col);
        xs_tbl->add_column(typ_col);
        xs_tbl->add_column(fr_loc_col);
        xs_tbl->add_column(to_loc_col);
        xs_tbl->add_column(y_col);
        xs_tbl->add_column(pred_col);

        Batch *test_batch = sup->create_batch();

        for (int i = 0; i < test_batch->get_size(); i++)
        {
            Tensor *pred = lm->forward(test_batch->get_x(i), false);
            pred_col->set_val(i, pred->get_val(0));
            delete pred;
        }

        delete test_batch;

        Table::to_csv("temp/preds.csv", xs_tbl);
    }

    // ===================================================================================================

    // Grad Check:
    {
        // Batch *grad_check_batch = sup->create_batch();
        // lm->grad_check(grad_check_batch->get_x(1), grad_check_batch->get_y(1), false);
        // delete grad_check_batch;
    }

    return 0;
}