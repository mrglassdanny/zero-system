#include "hip/hip_runtime.h"

#include <zero_system/mod.cuh>

#define ACTCODTYP_EMBG_DIM_CNT 4
#define LOC_EMBG_DIM_CNT 12

std::vector<int> get_output_shape()
{
    std::vector<int> n_shape{1};
    return n_shape;
}

void forward(Tensor *n, Tensor *nxt_n, bool train_flg)
{
    // t = aq + c + v

    float a = n->get_val(0);
    float c = n->get_val(1);
    float q = n->get_val(2);
    float v = 0.0f;

    int src_loc_beg_idx = 3;
    int dst_loc_beg_idx = src_loc_beg_idx + LOC_EMBG_DIM_CNT;

    for (int i = 0; i < LOC_EMBG_DIM_CNT; i++)
    {
        float loc_diff = n->get_val(src_loc_beg_idx + i) - n->get_val(dst_loc_beg_idx + i);
        v += (loc_diff * loc_diff);
    }

    v = sqrt(v);

    float t = a * q + c + v;

    nxt_n->set_val(0, t);
}

void forward2(Tensor *n, Tensor *nxt_n, bool train_flg)
{
    // t = a + v

    float a = n->get_val(0);
    float v = 0.0f;

    int src_loc_beg_idx = 1;
    int dst_loc_beg_idx = src_loc_beg_idx + LOC_EMBG_DIM_CNT;

    for (int i = 0; i < LOC_EMBG_DIM_CNT; i++)
    {
        float loc_diff = n->get_val(src_loc_beg_idx + i) - n->get_val(dst_loc_beg_idx + i);
        v += (loc_diff * loc_diff);
    }

    v = sqrt(v);

    float t = a + v;

    nxt_n->set_val(0, t);
}

Tensor *backward(Tensor *n, Tensor *dc)
{
    // t = aq + c + v

    Tensor *nxt_dc = new Tensor(n->get_device(), n->get_shape());

    float dc_val = dc->get_val(0);

    nxt_dc->set_val(0, dc_val * n->get_val(2));
    nxt_dc->set_val(1, dc_val * 1.0f);
    nxt_dc->set_val(2, dc_val * n->get_val(0));

    int src_loc_beg_idx = 3;
    int dst_loc_beg_idx = src_loc_beg_idx + LOC_EMBG_DIM_CNT;

    float v = 0.0f;

    for (int i = 0; i < LOC_EMBG_DIM_CNT; i++)
    {
        float loc_diff = n->get_val(src_loc_beg_idx + i) - n->get_val(dst_loc_beg_idx + i);
        v += (loc_diff * loc_diff);
    }

    v = sqrt(v);

    for (int i = 0; i < LOC_EMBG_DIM_CNT; i++)
    {
        if (v == 0.0f)
        {
            nxt_dc->set_val(src_loc_beg_idx + i, 0.0f);
            nxt_dc->set_val(dst_loc_beg_idx + i, 0.0f);
        }
        else
        {
            float dv = 1.0f / (2.0f * v);

            nxt_dc->set_val(src_loc_beg_idx + i, dc_val * dv * (2.0f * (n->get_val(src_loc_beg_idx + i) - n->get_val(dst_loc_beg_idx + i))));
            nxt_dc->set_val(dst_loc_beg_idx + i, dc_val * dv * (-2.0f * (n->get_val(src_loc_beg_idx + i) - n->get_val(dst_loc_beg_idx + i))));
        }
    }

    return nxt_dc;
}

Tensor *backward2(Tensor *n, Tensor *dc)
{
    // t = a + v

    Tensor *nxt_dc = new Tensor(n->get_device(), n->get_shape());

    float dc_val = dc->get_val(0);

    nxt_dc->set_val(0, dc_val * 1.0f);

    int src_loc_beg_idx = 1;
    int dst_loc_beg_idx = src_loc_beg_idx + LOC_EMBG_DIM_CNT;

    float v = 0.0f;

    for (int i = 0; i < LOC_EMBG_DIM_CNT; i++)
    {
        float loc_diff = n->get_val(src_loc_beg_idx + i) - n->get_val(dst_loc_beg_idx + i);
        v += (loc_diff * loc_diff);
    }

    v = sqrt(v);

    for (int i = 0; i < LOC_EMBG_DIM_CNT; i++)
    {
        if (v == 0.0f)
        {
            nxt_dc->set_val(src_loc_beg_idx + i, 0.0f);
            nxt_dc->set_val(dst_loc_beg_idx + i, 0.0f);
        }
        else
        {
            float dv = 1.0f / (2.0f * v);

            nxt_dc->set_val(src_loc_beg_idx + i, dc_val * dv * (2.0f * (n->get_val(src_loc_beg_idx + i) - n->get_val(dst_loc_beg_idx + i))));
            nxt_dc->set_val(dst_loc_beg_idx + i, dc_val * dv * (-2.0f * (n->get_val(src_loc_beg_idx + i) - n->get_val(dst_loc_beg_idx + i))));
        }
    }

    return nxt_dc;
}

void upd_rslt_fn(Tensor *p, Tensor *y, int *cnt)
{
    float y_val = y->get_val(0);
    float p_val = p->get_val(0);

    float lower = y_val < p_val ? y_val : p_val;
    float upper = y_val < p_val ? p_val : y_val;

    float prcnt = 1.0f - (lower / upper);

    if (prcnt <= 0.20f)
    {
        (*cnt)++;
    }
}

std::vector<float> loc_encode_fn(const char *loc_name, int dim_cnt)
{
    char delims[] = {'-'};
    char numerics[] = {'0', '1', '2', '3', '4', '5', '6', '7', '8', '9'};

    StackBuffer buf;

    std::vector<float> parsed_loc;

    int loc_name_len = strlen(loc_name);

    bool delim_flg = false;
    bool numeric_flg = false;
    bool alpha_flg = false;

    for (int i = 0; i < loc_name_len; i++)
    {
        char c = loc_name[i];

        for (int j = 0; j < sizeof(delims); j++)
        {
            if (c == delims[j])
            {
                if (buf.get_size() > 0)
                {
                    parsed_loc.push_back(atof(buf.get()));
                }

                buf.clear();

                delim_flg = true;
                numeric_flg = false;
                alpha_flg = false;

                break;
            }
        }

        if (!delim_flg)
        {
            numeric_flg = false;

            for (int j = 0; j < sizeof(numerics); j++)
            {
                if (c == numerics[j])
                {
                    numeric_flg = true;

                    if (alpha_flg)
                    {
                        if (buf.get_size() > 0)
                        {
                            parsed_loc.push_back(atof(buf.get()));
                        }

                        buf.clear();
                        buf.append(c);
                    }
                    else
                    {
                        buf.append(c);
                    }

                    alpha_flg = false;

                    break;
                }
            }
        }

        if (!delim_flg && !numeric_flg)
        {
            if (alpha_flg)
            {
                float buf_num = 0.0f;

                if (buf.get_size() > 0)
                {
                    buf_num = atof(buf.get());
                }

                buf.clear();
                buf_num += (int)c;
                buf.append(buf_num);
            }
            else
            {
                if (buf.get_size() > 0)
                {
                    parsed_loc.push_back(atof(buf.get()));
                }

                buf.clear();
                buf.append((int)c);
            }

            alpha_flg = true;
        }
    }

    if (buf.get_size() > 0)
    {
        parsed_loc.push_back(atof(buf.get()));
    }

    if (parsed_loc.size() < dim_cnt)
    {
        for (int i = parsed_loc.size(); i < dim_cnt; i++)
        {
            parsed_loc.push_back(0.0f);
        }
    }

    return parsed_loc;
}

// Using this fn will cause memory leak for child models!
Model *load_lm()
{
    Model *lm = new Model();
    lm->load("temp/lm.model");

    Model *actcodtyp_model = new Model();
    actcodtyp_model->load("temp/actcodtyp.model");

    Model *src_loc_model = new Model();
    src_loc_model->load("temp/loc.model");

    Model *dst_loc_model = new Model();
    dst_loc_model->load("temp/loc.model");
    dst_loc_model->share_parameters(src_loc_model);

    lm->child(actcodtyp_model);
    lm->child(src_loc_model);
    lm->child(dst_loc_model);

    // ((CustomLayer *)lm->get_layers()[0])->set_callbacks(get_output_shape, forward2, backward2);

    return lm;
}

void test(Supervisor *sup, Column *pred_col)
{
    Model *lm = load_lm();

    Batch *test_batch = sup->create_batch();
    lm->test(test_batch, upd_rslt_fn).print();

    for (int i = 0; i < test_batch->get_size(); i++)
    {
        Tensor *pred = lm->forward(test_batch->get_x(i), false);
        pred_col->set_val(i, pred->get_val(0));
        delete pred;
    }

    delete test_batch;

    delete lm;
}

void grad_check(Table *xs_tbl, Table *ys_tbl, Supervisor *sup)
{
    Model *lm = load_lm();

    Batch *grad_check_batch = sup->create_batch();

    lm->grad_check(grad_check_batch->get_x(1), grad_check_batch->get_y(1), true);

    delete grad_check_batch;

    delete lm;
}

int main(int argc, char **argv)
{
    ZERO();

    // Data setup:

    Table *xs_tbl = Table::fr_csv("data/palmov_data-test.csv");
    Table *ys_tbl = xs_tbl->split("elapsed_secs");

    Table *actcodtyps_tbl = Table::fr_csv("data/actcodtyps.csv");
    std::map<std::string, int> *actcodtyp_map = actcodtyps_tbl->get_column(0)->to_ordinal_map();

    Table *locs_tbl = Table::fr_csv("data/locs.csv");
    std::map<std::string, int> *loc_map = locs_tbl->get_column(0)->to_ordinal_map();

    Column *actcodtyp_col = new Column(*xs_tbl->get_column("actcodtyp"));
    Column *fr_loc_col = new Column(*xs_tbl->get_column("fr_loc"));
    Column *to_loc_col = new Column(*xs_tbl->get_column("to_loc"));

    delete xs_tbl->remove_column("actcod");
    delete xs_tbl->remove_column("typ");

    xs_tbl->encode_ordinal("actcodtyp", actcodtyp_map);
    xs_tbl->encode_ordinal("fr_loc", loc_map);
    xs_tbl->encode_ordinal("to_loc", loc_map);

    ys_tbl->scale_down();

    xs_tbl->print();

    Supervisor *sup;
    {
        Tensor *xs = Table::to_tensor(xs_tbl);
        Tensor *ys = Table::to_tensor(ys_tbl);

        Tensor::to_file("temp/xs.tr", xs);
        Tensor::to_file("temp/ys.tr", ys);

        std::vector<int> x_shape{xs->get_shape()[1]};
        sup = new Supervisor("temp/xs.tr", "temp/ys.tr", x_shape, 0);

        delete xs;
        delete ys;
    }

    // Fit:
    {
        Model *lm = new Model(0.1f);

        Model *actcodtyp_model = new Model();
        actcodtyp_model->embedding((int)actcodtyps_tbl->get_column(0)->row_cnt, ACTCODTYP_EMBG_DIM_CNT);

        Model *src_loc_model = new Model();
        src_loc_model->embedding((int)locs_tbl->get_column(0)->row_cnt, LOC_EMBG_DIM_CNT);

        Model *dst_loc_model = new Model();
        dst_loc_model->copy(src_loc_model);
        dst_loc_model->share_parameters(src_loc_model);

        lm->child(actcodtyp_model, xs_tbl->get_column_range("actcodtyp"));
        lm->child(src_loc_model, xs_tbl->get_column_range("fr_loc"));
        lm->child(dst_loc_model, xs_tbl->get_column_range("to_loc"));

        // lm->custom(lm->calc_adjusted_input_shape(xs_tbl->get_column_cnt()),
        //            get_output_shape, forward2, backward2);
        // lm->activation(ReLU);

        lm->dense(lm->calc_adjusted_input_shape(xs_tbl->get_column_cnt()), 256);
        lm->activation(ReLU);
        lm->dense(64);
        lm->activation(ReLU);
        lm->dense(1);

        lm->fit(sup, 25, 10, "temp/train.csv", upd_rslt_fn);

        Batch *test_batch = sup->create_batch();
        lm->test(test_batch, upd_rslt_fn).print();
        delete test_batch;

        lm->save("temp/lm.model");
        actcodtyp_model->save("temp/actcodtyp.model");
        src_loc_model->save("temp/loc.model");

        delete lm;
        delete actcodtyp_model;
        delete src_loc_model;
        delete dst_loc_model;
    }

    // Test:
    {
        Column *y_col = ys_tbl->get_column("elapsed_secs");
        Column *pred_col = new Column("pred", true, xs_tbl->get_row_cnt());

        xs_tbl->clear();

        xs_tbl->add_column(actcodtyp_col);
        xs_tbl->add_column(fr_loc_col);
        xs_tbl->add_column(to_loc_col);
        xs_tbl->add_column(y_col);
        xs_tbl->add_column(pred_col);

        test(sup, pred_col);

        Table::to_csv("temp/preds.csv", xs_tbl);
    }

    // Grad Check:
    {
        // grad_check(xs_tbl, ys_tbl, sup);
    }

    // Cleanup:

    delete sup;
    delete locs_tbl;

    return 0;
}